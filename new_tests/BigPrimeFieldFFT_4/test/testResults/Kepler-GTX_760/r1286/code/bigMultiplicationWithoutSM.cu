#include "hip/hip_runtime.h"
/* This file is part of the CUMODP library

    CUMODP is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    CUMODP is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with CUMODP.  If not, see <http://www.gnu.org/licenses/>.

    Copyright: Sardar Anisul Haque <shaque4@uwo.ca>
               Xin Li <xli.software@gmail.com>
               Farnam Mansouri <mansouri.farnam@gmail.com>
               Davood Mohajerani <dmohajer@uwo.ca>
               Marc Moreno Maza  <moreno@csd.uwo.ca>
               Wei Pan <wei.pan@intel.com>
               Ning Xie <nxie6@csd.uwo.ca>
*/


#include <time.h>
#include <stdio.h>
#include <unistd.h>

//BN block number
#define BN 256
//TN thread number in a block
#define TN 256 
//R  2^63+2^34
#define R 9223372054034644992
//RC R complement  RC=2^64-R
#define RC 9223372019674906624
//ULMAX=2^64-1
#define ULMAX 18446744073709551615
//sqrt(R) 3037000502
#define SQRTR 3037000502

__device__ void bigAdd2(unsigned long *xs, unsigned long *ys) 
{
  unsigned long us[8];
  short i, pos;
  unsigned short c=0;
  unsigned long num1,num2;
  num1=0;
  num2=R-1;
  
  for(i=0;i<=7;i++)
  {
    num1=xs[i]+ys[i]+c;
    if(num1<xs[i]||num1<ys[i]) //there is overflow/truncation
    {
      c=1;
      us[i]=num1+RC;
    }
    else if(num1>=R)
    {  
      c=1;
      us[i]=num1-R;
    }
    else
    {
      c=0;
      us[i]=num1;
    }
  }
  if(c>0)
  {
    pos=-1;
    for(i=0;i<8;i++) 
    {
      if(us[i] != 0)
      {
        pos=i;
        break;
      }
    }
    if(pos>=0)
    {
      for(i=0;i<pos;i++)
      {
          us[i] = num2;
      }
      us[pos] = us[pos] - 1;
    }            
    else
    {
      us[0]=ULMAX;
      for(i=1;i<8;i++)
      {
        us[i]=0;
      }
    }  
  }  
  xs[0]=us[0];
  xs[1]=us[1];
  xs[2]=us[2];
  xs[3]=us[3];
  xs[4]=us[4];
  xs[5]=us[5];
  xs[6]=us[6];
  xs[7]=us[7];
}


__device__ void bigSub2(unsigned long *xs, unsigned long *ys) 
{  
  unsigned long us[8];
  short i, pos;
  unsigned short c=0;
  unsigned long num1,num2;
  num1=0;
  num2=R-1;
  
  for(i=0;i<=7;i++)
  {
    num1=ys[i]+c;
    if(xs[i]<num1) //there is not enough to do subtraction
    {
      c=1;
      us[i]=R-num1+xs[i];
    }
    else
    {
      c=0;
      us[i]=xs[i]-num1;
    }
  }
  if(c>0)
  {
    pos=-1;
    for(i=0;i<8;i++) 
    {
      if(us[i] < num2)
      {
        pos=i;
        break;
      }
    }
    if(pos>=0)
    {
      for(i=0;i<pos;i++)
      {
          us[i] = 0;
      }
      us[pos] ++;
    }            
    else
    {
      us[0]=ULMAX;
      for(i=1;i<8;i++)
      {
        us[i]=0;
      }
    }  
  }
  xs[0]=us[0];
  xs[1]=us[1];
  xs[2]=us[2];
  xs[3]=us[3];
  xs[4]=us[4];
  xs[5]=us[5];
  xs[6]=us[6];
  xs[7]=us[7];  
}


//[l,h,c]
__device__ void mulLong(unsigned long x, unsigned long y, unsigned long *s)
{
  short x1,y1;
  unsigned long l,h,c,x0,y0,v2,v5,v9,v10,v11,v14,v15,v16,v17,q,t;
  unsigned long a0,a1,b0,b1,c0,c1,c1prime,d0,d1,d2,e0,e1;
  
  if(x<=SQRTR && y<=SQRTR)
  {
    s[0]=x*y;
    s[1]=0;
    s[2]=0;
    return;
  }
  
  x1=(x>=R?1:0);
  x0=(x1>0?x-R:x);
  y1=(y>=R?1:0);
  y0=(y1>0?y-R:y);
  
  //x1,y1 = {0,1}
  //v2=x0*y1; //[0,v2,0];
  v2=(y1>0)?x0:0;
  //v5=x1*y0; //[0,v5,0];
  v5=(x1>0)?y0:0;
  //v9=x1*y1; //[0,0,1];
  v9=(x1+y1>2)?1:0;
  
  c=v9;
  l=0;
  h=v5+v2;
  h<v5||h<v2?(c=c+1):(c=c);
  c>v9?(h=h+RC):(h=h);
  
  if(x0<=SQRTR&&y0<=SQRTR)
  {
    s[0]=x0*y0;
    s[1]=h;
    s[2]=c;
    return;
  }
  
  //lhc
  //x0*y0
  a1=x0>>32;
  a0=x0-(a1<<32);
  b1=y0>>32;
  b0=y0-(b1<<32);
  
  c0=0;
  c1=a1*b1;
  
  t=a0*b1;
  q=t>>32;
  t=(t-(q<<32))<<32;
  c1+=q;
  c0+=t;  //safe
  
  t=a1*b0;
  q=t>>32;
  t=(t-(q<<32))<<32;
  c1+=q;
  q=c0+t;               //here, is not related to r.
  q<c0||q<t?(c1++):(c1=c1);  //c0=c0+t and carry, safe
  c0=q;
  
  t=a0*b0;
  q=c0+t;
  q<c0||q<t?(c1++):(c1=c1);  //Now we finish [c0,c1]=x0*y0
  c0=q;
  
  c1prime=c1<<1;
  
  c0>=R?(v11=1):(v11=0);
  v11>0?(v10=c0-R):(v10=c0);
  //v12=0;
  
  q=l+v10;  //[l,h,c] + [v10,v11,0]
  q<l||q<v10?(v11=v11+1):(v11=v11);
  q<l||q<v10?(l=q+RC):(l=q);
  if(l>=R)
  {
    l=l-R;
    v11++;
  }
  q=h+v11;
  q<h||q<v11?(c=c+1):(c=c);
  q<h||q<v11?(h=q+RC):(h=q);
  if(h>=R)
  {
    h=h-R;
    c++;
  }
  //v13=0;
  c1prime>=R?(v15=1):(v15=0);
  v15>0?(v14=c1prime-R):(v14=c1prime); //v13=0;
  
  q=h+v14;  //[l,h,c]+[0,v14,v15]
  q<h||q<v14?(c=c+v15+1):(c=c+v15);
  q<h||q<v14?(h=q+RC):(h=q);
  if(h>=R)
  {
    h=h-R;
    c++;
  }
  //[l,h,c]
  
  d1=c1prime>>29;
  d0=c1prime-(d1<<29);
  if(d0>=d1)
  {
    d2=d0-d1;
    e1=d2>>29;
    e0=d2-(e1<<29);
    e0>=e1?(v16=(e0-e1)<<34):(v16=R-(e1<<34)+(e0<<34));
    e0>=e1?(v17=e1+d1):(v17=e1+d1-1);
    /*
    if(e0>=e1)
    {
      v16=(e0-e1)<<34;
      v17=e1+d1;
    }
    else
    {
      v17=e1+d1-1;
      v16=R-(e1<<34)+(e0<<34);
    }
    */
  }
  else
  {
    //d1>d0
    d2=d1-d0;
    e1=d2>>29;
    e0=d2-(e1<<29);
    e0>=e1?(v16=R-((e0-e1)<<34)):(v16=(e1-e0)<<34);
    e0>=e1?(v17=d1-e1-1):(v17=d1-e1);
    /*
    if(e0>=e1)
    {
      v16=R-((e0-e1)<<34);
      v17=d1-e1-1;
    }
    else
    {
      v16=(e1-e0)<<34;
      v17=d1-e1;
    }
    */
  }
  //[l,h,c]-[v16,v17,0]
  //q
  q=0;
  if(l>=v16)
  {
    l=l-v16;
  }
  else
  {
    l=R-v16+l;
    q=1;
  }
  //t
  if(h<q+v17)
  {
    c=c-1;
    h=R-q-v17+h;
  }
  else
  {
    h=h-q-v17;
  }  
  s[0]=l;
  s[1]=h;
  s[2]=c;
}

//store in l0, h0, c0
__device__ void smallAdd(unsigned long *l0, unsigned long *h0, short *c0, unsigned long *l1, unsigned long *h1, unsigned long *c1)
{
  short c=0;
  unsigned long s=0;
  s=*l0+*l1;
  s<*l0 || s<*l1?c=1:c=0;
  c>0?s=s+RC:s=s;
  if(s>=R)
  {
    s=s-R;
    c=1;
  }
  *l0=s;
  
  *h1=*h1+c;  //h1<r<2^64-1. This means no overflow
  s=*h0+*h1;
  s<*h0||s<*h1?c=1:c=0;
  c>0?s=s+RC:s=s;
  if(s>=R)
  {
    s=s-R;
    c=1;
  }
  *h0=s;
  
  *c0=*c0+(short)*c1+c;
}

__device__ void bigMul(unsigned long *xs, unsigned long *ys, unsigned long *us) 
{
    unsigned long ts1[8];
    unsigned long ts2[8];
    unsigned long rs[3];
    short c0,c1,c2,c3,c4,c5,c6,c7;
    unsigned long l0,l1,l2,l3,l4,l5,l6,l7,h0,h1,h2,h3,h4,h5,h6,h7;
    
    //x0*y0
    mulLong(xs[0],ys[0],rs);
    l0=rs[0];
    h0=rs[1];
    c0=(short)rs[2];
    
    //x0*y1+x1*y0
    mulLong(xs[0],ys[1],rs);    
    l1=rs[0];
    h1=rs[1];
    c1=(short)rs[2];
    mulLong(xs[1],ys[0],rs);
    smallAdd(&l1,&h1,&c1,&rs[0],&rs[1],&rs[2]);
    
    //x0*y2+x1*y1+x2*y0
    mulLong(xs[0],ys[2],rs);    
    l2=rs[0];
    h2=rs[1];
    c2=(short)rs[2];
    mulLong(xs[1],ys[1],rs);
    smallAdd(&l2,&h2,&c2,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[2],ys[0],rs);
    smallAdd(&l2,&h2,&c2,&rs[0],&rs[1],&rs[2]);
    
    //x0*y3+x1*y2+x2*y1+x3*y0
    mulLong(xs[0],ys[3],rs);    
    l3=rs[0];
    h3=rs[1];
    c3=(short)rs[2];
    mulLong(xs[1],ys[2],rs);
    smallAdd(&l3,&h3,&c3,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[2],ys[1],rs);
    smallAdd(&l3,&h3,&c3,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[3],ys[0],rs);
    smallAdd(&l3,&h3,&c3,&rs[0],&rs[1],&rs[2]);    
    
    //x0*y4+x1*y3+x2*y2+x3*y1+x4*y0
    mulLong(xs[0],ys[4],rs);    
    l4=rs[0];
    h4=rs[1];
    c4=(short)rs[2];
    mulLong(xs[1],ys[3],rs);
    smallAdd(&l4,&h4,&c4,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[2],ys[2],rs);
    smallAdd(&l4,&h4,&c4,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[3],ys[1],rs);
    smallAdd(&l4,&h4,&c4,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[4],ys[0],rs);
    smallAdd(&l4,&h4,&c4,&rs[0],&rs[1],&rs[2]);
    
    //x0*y5+x1*y4+x2*y3+x3*y2+x4*y1+x5*y0
    mulLong(xs[0],ys[5],rs);    
    l5=rs[0];
    h5=rs[1];
    c5=(short)rs[2];
    mulLong(xs[1],ys[4],rs);
    smallAdd(&l5,&h5,&c5,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[2],ys[3],rs);
    smallAdd(&l5,&h5,&c5,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[3],ys[2],rs);
    smallAdd(&l5,&h5,&c5,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[4],ys[1],rs);
    smallAdd(&l5,&h5,&c5,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[5],ys[0],rs);
    smallAdd(&l5,&h5,&c5,&rs[0],&rs[1],&rs[2]);
 
    //x0*y6+x1*y5+x2*y4+x3*y3+x4*y2+x5*y1+x6*y0
    mulLong(xs[0],ys[6],rs);    
    l6=rs[0];
    h6=rs[1];
    c6=(short)rs[2];
    mulLong(xs[1],ys[5],rs);
    smallAdd(&l6,&h6,&c6,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[2],ys[4],rs);
    smallAdd(&l6,&h6,&c6,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[3],ys[3],rs);
    smallAdd(&l6,&h6,&c6,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[4],ys[2],rs);
    smallAdd(&l6,&h6,&c6,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[5],ys[1],rs);
    smallAdd(&l6,&h6,&c6,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[6],ys[0],rs);
    smallAdd(&l6,&h6,&c6,&rs[0],&rs[1],&rs[2]);
    
    //x0*y7+x1*y6+x2*y5+x3*y4+x4*y3+x5*y2+x6*y1+x7*y0
    mulLong(xs[0],ys[7],rs);    
    l7=rs[0];
    h7=rs[1];
    c7=(short)rs[2];
    mulLong(xs[1],ys[6],rs);
    smallAdd(&l7,&h7,&c7,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[2],ys[5],rs);
    smallAdd(&l7,&h7,&c7,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[3],ys[4],rs);
    smallAdd(&l7,&h7,&c7,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[4],ys[3],rs);
    smallAdd(&l7,&h7,&c7,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[5],ys[2],rs);
    smallAdd(&l7,&h7,&c7,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[6],ys[1],rs);
    smallAdd(&l7,&h7,&c7,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[7],ys[0],rs);
    smallAdd(&l7,&h7,&c7,&rs[0],&rs[1],&rs[2]);
    
    // (c5+h6+l7)*r^7+(c4+h5+l6)*r^6+(c3+h4+l5)*r^5+(c2+h3+l4)*r^4+(c1+h2+l3)*r^3+(c0+h1+l2)*r^2+(h0+l1-c7)*r+(l0-c6-h7)
    ts1[0]=l0;ts1[1]=h0;ts1[2]=c0;ts1[3]=c1;
    ts1[4]=c2;ts1[5]=c3;ts1[6]=c4;ts1[7]=c5;
    ts2[0]=0;ts2[1]=l1;ts2[2]=h1;ts2[3]=h2;
    ts2[4]=h3;ts2[5]=h4;ts2[6]=h5;ts2[7]=h6;
    bigAdd2(ts1,ts2);
    ts2[0]=0;ts2[1]=0;ts2[2]=l2;ts2[3]=l3;
    ts2[4]=l4;ts2[5]=l5;ts2[6]=l6;ts2[7]=l7;
    bigAdd2(ts1,ts2);
    ts2[0]=c6;ts2[1]=c7;ts2[2]=0;ts2[3]=0;
    ts2[4]=0;ts2[5]=0;ts2[6]=0;ts2[7]=0;
    bigSub2(ts1,ts2);
    ts2[0]=h7;ts2[1]=0;ts2[2]=0;ts2[3]=0;
    ts2[4]=0;ts2[5]=0;ts2[6]=0;ts2[7]=0;
    bigSub2(ts1,ts2);
    
    //(x7*y7)r^6
    mulLong(xs[7],ys[7],rs);
    l6=rs[0];
    h6=rs[1];
    c6=(short)rs[2];
    
    //(x6*y7+x7*y6)r^5
    mulLong(xs[6],ys[7],rs);    
    l5=rs[0];
    h5=rs[1];
    c5=(short)rs[2];
    mulLong(xs[7],ys[6],rs);
    smallAdd(&l5,&h5,&c5,&rs[0],&rs[1],&rs[2]);
    
    //(x5*y7+x6*y6+x7*y5)r^4
    mulLong(xs[5],ys[7],rs);    
    l4=rs[0];
    h4=rs[1];
    c4=(short)rs[2];
    mulLong(xs[6],ys[6],rs);
    smallAdd(&l4,&h4,&c4,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[7],ys[5],rs);
    smallAdd(&l4,&h4,&c4,&rs[0],&rs[1],&rs[2]);    
    
    //(x4*y7+x5*y6+x6*y5+x7*y4)r^3
    mulLong(xs[4],ys[7],rs);    
    l3=rs[0];
    h3=rs[1];
    c3=(short)rs[2];
    mulLong(xs[5],ys[6],rs);
    smallAdd(&l3,&h3,&c3,&rs[0],&rs[1],&rs[2]);    
    mulLong(xs[6],ys[5],rs);
    smallAdd(&l3,&h3,&c3,&rs[0],&rs[1],&rs[2]);    
    mulLong(xs[7],ys[4],rs);
    smallAdd(&l3,&h3,&c3,&rs[0],&rs[1],&rs[2]);
    
    //(x3*y7+x4*y6+x5*y5+x6*y4+x7*y3)r^2
    mulLong(xs[3],ys[7],rs);    
    l2=rs[0];
    h2=rs[1];
    c2=(short)rs[2];
    mulLong(xs[4],ys[6],rs);
    smallAdd(&l2,&h2,&c2,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[5],ys[5],rs);
    smallAdd(&l2,&h2,&c2,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[6],ys[4],rs);
    smallAdd(&l2,&h2,&c2,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[7],ys[3],rs);
    smallAdd(&l2,&h2,&c2,&rs[0],&rs[1],&rs[2]);
    
    //(x2*y7+x3*y6+x4*y5+x5*y4+x6*y3+x7*y2)r
    mulLong(xs[2],ys[7],rs);    
    l1=rs[0];
    h1=rs[1];
    c1=(short)rs[2];
    mulLong(xs[3],ys[6],rs);
    smallAdd(&l1,&h1,&c1,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[4],ys[5],rs);
    smallAdd(&l1,&h1,&c1,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[5],ys[4],rs);
    smallAdd(&l1,&h1,&c1,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[6],ys[3],rs);
    smallAdd(&l1,&h1,&c1,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[7],ys[2],rs);
    smallAdd(&l1,&h1,&c1,&rs[0],&rs[1],&rs[2]);
    
    //(x1*y7+x2*y6+x3*y5+x4*y4+x5*y3+x6*y2+x7*y1)
    mulLong(xs[1],ys[7],rs);    
    l0=rs[0];
    h0=rs[1];
    c0=(short)rs[2];
    mulLong(xs[2],ys[6],rs);
    smallAdd(&l0,&h0,&c0,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[3],ys[5],rs);
    smallAdd(&l0,&h0,&c0,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[4],ys[4],rs);
    smallAdd(&l0,&h0,&c0,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[5],ys[3],rs);
    smallAdd(&l0,&h0,&c0,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[6],ys[2],rs);
    smallAdd(&l0,&h0,&c0,&rs[0],&rs[1],&rs[2]);
    mulLong(xs[7],ys[1],rs);
    smallAdd(&l0,&h0,&c0,&rs[0],&rs[1],&rs[2]);
    
    //(c5+h6)*r^7+(c4+h5+l6)*r^6+(c3+h4+l5)*r^5+(c2+h3+l4)*r^4+(c1+h2+l3)*r^3+(c0+h1+l2)*r^2+(h0+l1)*r+(l0-c6)
    ts2[0]=l0;ts2[1]=h0;ts2[2]=c0;ts2[3]=c1;
    ts2[4]=c2;ts2[5]=c3;ts2[6]=c4;ts2[7]=c5;
    bigSub2(ts1,ts2);
    ts2[0]=0;ts2[1]=l1;ts2[2]=h1;ts2[3]=h2;
    ts2[4]=h3;ts2[5]=h4;ts2[6]=h5;ts2[7]=h6;
    bigSub2(ts1,ts2);
    ts2[0]=0;ts2[1]=0;ts2[2]=l2;ts2[3]=l3;
    ts2[4]=l4;ts2[5]=l5;ts2[6]=l6;ts2[7]=0;
    bigSub2(ts1,ts2);
    ts2[0]=c6;ts2[1]=0;ts2[2]=0;ts2[3]=0;
    ts2[4]=0;ts2[5]=0;ts2[6]=0;ts2[7]=0;
    bigAdd2(ts1,ts2);
    us[0]=ts1[0];
    us[1]=ts1[1];
    us[2]=ts1[2];
    us[3]=ts1[3];
    us[4]=ts1[4];
    us[5]=ts1[5];
    us[6]=ts1[6];
    us[7]=ts1[7];
}


__global__ void batchBigMul(unsigned long r, unsigned long *xs, unsigned long *ys, unsigned long *us) 
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;   
  short t;
  unsigned long *xd, *yd, *ud; 
  
  
  xd = (unsigned long*)((char*)xs + tid*sizeof(unsigned long)*8);
  yd = (unsigned long*)((char*)ys + tid*sizeof(unsigned long)*8);
  ud = (unsigned long*)((char*)us + tid*sizeof(unsigned long)*8);
  __syncthreads();
  
  for(t=0;t<100;t++)
  {
    bigMul(xd,yd,ud);    
  }
}

int main(int argc, char *argv[])
{
  char fileName[1024];
	FILE *fp1;	 
  unsigned long *xs, *ys, *us, *xs_d, *ys_d, *us_d; 
  hipEvent_t start, stop;
  float elapsedTime;
  
  xs=(unsigned long *)malloc((sizeof(unsigned long)*8*TN*BN));
  ys=(unsigned long *)malloc((sizeof(unsigned long)*8*TN*BN));
  us=(unsigned long *)malloc((sizeof(unsigned long)*8*TN*BN));
  
  memset(fileName, 0 , sizeof(char)*1024);
  sprintf(fileName, "big_number_benchmark.dat");
  if((fp1=fopen(fileName,"rb"))==NULL)
  {
  	printf("fail to %s", fileName);
  	fclose(fp1);
  	exit(-1);
  } 
  
  //��ȡ���е����ݡ�
  memset(xs,0,sizeof(unsigned long)*TN*BN);
  fread(xs,sizeof(unsigned long),TN*BN,fp1);
  memset(ys,0,sizeof(unsigned long)*TN*BN);
  fread(ys,sizeof(unsigned long),TN*BN,fp1);
  
  
  //begin gpu
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  
  //init gpu memory
  hipMalloc((void **)&xs_d, sizeof(unsigned long)*8*TN*BN);
  hipMemcpy(xs_d, xs, sizeof(unsigned long)*8*TN*BN, hipMemcpyHostToDevice);
  hipMalloc((void **)&ys_d, sizeof(unsigned long)*8*TN*BN);
  hipMemcpy(ys_d, ys, sizeof(unsigned long)*8*TN*BN, hipMemcpyHostToDevice);
  hipMalloc((void **)&us_d, sizeof(unsigned long)*8*TN*BN);
  hipMemcpy(us_d, us, sizeof(unsigned long)*8*TN*BN, hipMemcpyHostToDevice);
  
  batchBigMul<<<BN,TN>>>(R,xs_d,ys_d,us_d);
  hipDeviceSynchronize();
  hipMemcpy(us, us_d, sizeof(unsigned long)*8*TN*BN, hipMemcpyDeviceToHost);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  
  printf("we have done 100 times of us = xs * ys with shared memory, %d big number pair.\n", BN*TN);
  printf("the time of gpu is %f ms\n", elapsedTime);
  
  //free
  fclose(fp1);
  hipFree(xs_d);	
  hipFree(ys_d);	
  hipFree(us_d);	
  free(xs);
  free(ys);
  free(us);
}



