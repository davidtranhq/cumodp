#include "hip/hip_runtime.h"
/* This file is part of the CUMODP library

    CUMODP is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    CUMODP is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with CUMODP.  If not, see <http://www.gnu.org/licenses/>.

    Copyright: Sardar Anisul Haque <shaque4@uwo.ca>
               Xin Li <xli.software@gmail.com>
               Farnam Mansouri <mansouri.farnam@gmail.com>
               Davood Mohajerani <dmohajer@uwo.ca>
               Marc Moreno Maza  <moreno@csd.uwo.ca>
               Wei Pan <wei.pan@intel.com>
               Ning Xie <nxie6@csd.uwo.ca>
*/


#include <time.h>
#include <stdio.h>
#include <unistd.h>

__global__ void batchBigSub(unsigned long r, unsigned long *xs, unsigned long *ys, unsigned long *us) 
{  
  int tid = threadIdx.x;  
  short i, pos;
  unsigned short c=0;
  unsigned long num1,num2;
  unsigned long *xd, *yd, *ud; 
  num1=0;
  num2=r-1;
  
  xd = (unsigned long*)((char*)xs + tid*sizeof(unsigned long)*8);
  yd = (unsigned long*)((char*)ys + tid*sizeof(unsigned long)*8);
  ud = (unsigned long*)((char*)us + tid*sizeof(unsigned long)*8);

  
  for(i=0;i<=7;i++)
  {
    num1=yd[i]+c;
    if(xd[i]<num1) //there is not enough to do subtraction
    {
      c=1;
      ud[i]=r-num1+xd[i];
    }
    else
    {
      c=0;
      ud[i]=xd[i]-num1;
    }
  }
  if(c>0)
  {
    pos=-1;
    for(i=0;i<8;i++) 
    {
      if(ud[i] < num2)
      {
        pos=i;
        break;
      }
    }
    if(pos>=0)
    {
      for(i=0;i<pos;i++)
      {
          ud[i] = 0;
      }
      ud[pos] ++;
    }            
    else
    {
      ud[0]=18446744073709551615u;
      for(i=1;i<8;i++)
      {
        ud[i]=0;
      }
    }  
  }
  
}

int main(int argc, char *argv[])
{
  int i,j;
  int L=16;  //L should be smaller than 1024.
  int pos=0;
  unsigned long r = 9223372054034644992;
  unsigned long *xs, *ys, *us, *xs_d, *ys_d, *us_d; 
  xs=(unsigned long *)malloc((sizeof(unsigned long)*8 * L));
  ys=(unsigned long *)malloc((sizeof(unsigned long)*8 * L));
  us=(unsigned long *)malloc((sizeof(unsigned long)*8 * L));
  //init value
  for(j=0;j<L;j++)
  {
    for(i=0;i<8;i++)
    {
      pos=j*8+i;  // pos++;
      xs[pos] = 1;
      ys[pos] = 9223372054034644990u;
      us[pos] = 0;
    }
  }
  
  //init gpu memory
  hipMalloc((void **)&xs_d, sizeof(unsigned long) * 8 * L);
  hipMemcpy(xs_d, xs, sizeof(unsigned long) * 8 * L, hipMemcpyHostToDevice);
  hipMalloc((void **)&ys_d, sizeof(unsigned long) * 8 * L);
  hipMemcpy(ys_d, ys, sizeof(unsigned long) * 8 * L, hipMemcpyHostToDevice);
  hipMalloc((void **)&us_d, sizeof(unsigned long) * 8 * L);
  hipMemcpy(us_d, us, sizeof(unsigned long) * 8 * L, hipMemcpyHostToDevice);
  
  batchBigSub<<<1,L>>>(r,xs_d,ys_d,us_d);
  hipDeviceSynchronize();
  hipMemcpy(us, us_d, sizeof(unsigned long) * 8 * L, hipMemcpyDeviceToHost);
  
  printf("we have done us = xs - ys.\n");
  printf("the xs is \n");
  for(i=0;i<16;i++)
  {
    printf ("%lu\n", xs[i]);
  }
  printf("the ys is \n");
  for(i=0;i<16;i++)
  {
    printf ("%lu\n", ys[i]);
  }
  printf("the us is \n");
  for(i=0;i<16;i++)
  {
    printf ("%lu\n", us[i]);
  }
  
}

__global__ void addition1(unsigned long *a) 
{
  *a = *a + 1;
}

int main1(int argc, char *argv[])
{	
  unsigned long *a;
  unsigned long *a_d;
  a=(unsigned long *)malloc((sizeof(unsigned long)*1));  
  a[0] = 18446744073709551614u;
  printf("a is %lu\n",a[0]);
  hipMalloc((void **)&a_d, sizeof(unsigned long) * 1);
  hipMemcpy(a_d, a, sizeof(unsigned long) * 1, hipMemcpyHostToDevice);
  addition1<<<1,1>>>(a_d);
  hipDeviceSynchronize();
  hipMemcpy(a, a_d, sizeof(unsigned long) * 1, hipMemcpyDeviceToHost);
  printf("a is %lu\n",a[0]);
  return 0;
}

