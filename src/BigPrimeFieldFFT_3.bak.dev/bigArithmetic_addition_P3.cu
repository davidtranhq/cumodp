#include "hip/hip_runtime.h"

#include "BigPrimeFieldFFT_3/bigPrimeField_P3.h"
//#include "BigPrimeFieldFFT_3/bigArithmetic_subtraction_P3.h"
//#include "BigPrimeFieldFFT_3/bigArithmetic_cyclicShift_P3.h"

/**********************************************/
/**********************************************/
__device__ __inline__ void
device_p3_bigPrimeSub_plain_ptx_v0_local_add (usfixn64 * __restrict__ xm,
	usfixn64 * __restrict__ ym)
{
	unsigned short c = 0;
//	short pos1;
	usfixn32 pos1;
	usfixn64 num1;

	usfixn32 bitsFlag = 0, bit = 0;

//	num1 = 0;
//	num2 = R - 1;
	short i;
	for (i = 0; i < 8; i++)
	{
		num1 = ym[i] + c;
		if (xm[i] < num1) //there is not enough to do subtraction
		{
			c = 1;
			xm[i] = R - num1 + xm[i];
		}
		else
		{
			c = 0;
			xm[i] = xm[i] - num1;
		}
		bit = (xm[i] <= R_MINUS_ONE);
		bitsFlag |= (bit << i);
	}

	asm("brev.b32 %0,%0;":"+r"(bitsFlag):);
	asm("bfind.u32 %0, %1;":"=r"(pos1):"r"(bitsFlag));
//	if (pos1 == 0xFFFFFFFF)
//		pos1 = -1;
//	else
//		pos1 = 31 - pos1;

	pos1 = 31 - pos1;
//	if (c > 0)
//	{
//		if (pos1 >= 0)
//		{
//			for (i = 0; i < pos1; i++)
//				xm[i] = 0;
//			xm[i]++;
//		}
//		else
//		{
//			xm[0] = ULMAX;
//			for (i = 1; i < 8; i++)
//				xm[i] = 0;
//		}
//	}

	if (c > 0 && pos1 < 32)
	{
		for (i = 0; i < pos1; i++)
			xm[i] = 0;
		xm[i]++;
	}
	if (c > 0 && pos1 == 32)
	{
		xm[0] = ULMAX;
		for (i = 1; i < 8; i++)
			xm[i] = 0;
	}
}

/**********************************************/
/**********************************************/
__device__ void
device_p3_bigSub_uVector_plain_2_local_add (uvector8 & __restrict__ xm,
	const uvector8 & __restrict__
	ym)
{
//	usfixn64 um[8];
	short i, posSub = -1;
	unsigned short c = 0;
	usfixn64 num1;
//	, num2;
	num1 = 0;
//	num2 = R - 1;

//#pragma unroll COEFFICIENT_SIZE
	for (i = 0; i < 8; i++)
	{
//		num1 = ym[i] + c;
		num1 = getUvector8Element (ym, i) + c;
//		if (xm[i] < num1) //there is not enough to do subtraction
		if (getUvector8Element (xm, i) < num1) //there is not enough to do subtraction
		{
			c = 1;
//			xm[i] = R - num1 + xm[i];
//			num1 = R - num1 + xm[i];
			num1 = R - num1 + getUvector8Element (xm, i);
		}
		else
		{
			c = 0;
//			xm[i] = xm[i] - num1;
//			num1 = xm[i] - num1;
			num1 = getUvector8Element (xm, i) - num1;
		}
//		xm[i] = num1;
		setUvector8Element (xm, i, num1);
		posSub = (posSub == -1 && num1 <= R_MINUS_ONE) * i;
	}

	if (c > 0 && posSub >= 0)
	{
		for (i = 0; i < posSub; i++)
		{
//			xm[i] = 0;
			setUvector8Element (xm, i, 0);
		}
//		memset(xm, 0x00, posSub*sizeof(usfixn64));
//		xm[posSub]++;
		setUvector8Element (xm, posSub, getUvector8Element (xm, posSub) + 1);
	}
	if (c > 0 && posSub < 0)
	{
//		xm[0] = ULMAX;
		setUvector8Element (xm, 0, ULMAX);
//#pragma unroll (COEFFICIENT_SIZE-1)
		for (i = 1; i < 8; i++)
		{
//			xm[i] = 0;
			setUvector8Element (xm, i, 0);
		}
	}

}

/**********************************************/
//working and faster than permutated_3 and permutated_4
__device__ void inline
device_p3_cyclicShift_permutated_5_local_add (usfixn64 * __restrict__ xs, const short sn,
	const usfixn64 permutationStride)
{

	if (sn <= 0 || sn > 8)
	{
		return;
	}
	short i = 0, j = 0;
//	usfixn64 ts[8] = { 0 };
//	usfixn64 ys[8] = { 0 };
	uConstArray8_align8 ts, ys;

//#pragma unroll COEFFICIENT_SIZE
	for (i = 0; i < 8; i++)
	{
		ts.i[i] = 0;
		ys.i[i] = 0;
	}

//	memset(ts.i,0x00, COEFFICIENT_SIZE*sizeof(usfixn64));
//	memset(ys.i,0x00, COEFFICIENT_SIZE*sizeof(usfixn64));

//	memset(ts.i,0x00,64);
//		memset(ys.i,0x00,64);

	j = 8 - sn;
	usfixn64 offset = j * permutationStride;
//	for (i = 0; i < sn; i++)
//	{
//		ts[i] = xs[j++];
//	}
//	for (i = 0; i < sn; i++)
//	{
////		ts[i] = xs[offset];
//		ts.i[i] = xs[offset];
//		offset += permutationStride;
//	}

	for (i = 0; i < sn; i++)
	{
		//		ts[i] = xs[offset];
		ts.i[i] = xs[offset];
		offset += permutationStride;
	}

	offset = (7 - sn) * permutationStride;
	j = 7;
	for (i = 7 - sn; i >= 0; i--)
	{
//		xs[i + sn] = xs[i];
//		ys[i + sn] = xs[offset];
//		ys.i[i + sn] = xs[offset];
		ys.i[j--] = xs[offset];
		offset -= permutationStride;
	}
//	for (i = 0; i < sn; i++)
//	{
//		xs[i] = 0;
//	}
//	device_p3_bigSub(ys, ts, xs);
//	device_bigSub_plain(&ys.i[0], &ts.i[0]);
//	device_bigSub_plain_1(&ys.i[0], &ts.i[0]);
	device_p3_bigPrimeSub_plain_ptx_v0_local_add (ys.i, ts.i);
//	device_bigPrimeSub_permutated_3(xs,ts, permutationStride);
	offset = 0;
//#pragma unroll COEFFICIENT_SIZE
	for (i = 0; i < 8; i++)
	{
//		xs[offset] = ys[i];
//		xs[offset] = ys.i[i];
		xs[offset] = ys.i[i];
		offset += permutationStride;
//		xs[offset8[i]] = ys.i[i];
	}
}

/**********************************************/
//computing cyclic shift with vectors of size 8 on register
__device__ __inline__ void
device_p3_cyclicShift_permutated_7_local_add (usfixn64 * __restrict__ xs, const short & sn,
	const usfixn64 & permutationStride)
{
	if (sn <= 0 || sn > 8)
	{
		return;
	}
	short i = 0, j = 0;
//	usfixn64 ts[8] = { 0 };
//	usfixn64 ys[8] = { 0 };
//	uConstArray8_align8 ts, ys;
	uvector8 ts, ys;
	usfixn64 offset = 0;
//#pragma unroll COEFFICIENT_SIZE
//	for(i=0;i<COEFFICIENT_SIZE;i++)
//	{
//		ys.i[i]=xs[offset];
//		ts.i[i]=0;
//		offset+=permutationStride;
//	}

//#pragma unroll COEFFICIENT_SIZE
//	for (i = 0; i < COEFFICIENT_SIZE; i++)
//	{
////		ys.i[i]=xs[offset];
//		setUvector8Element(ys, i, xs[offset]);
////		ts.i[i]=0;
//		setUvector8Element(ts, i, 0);
//		offset += permutationStride;
//	}

//#pragma unroll COEFFICIENT_SIZE
//	for (i = 0; i < COEFFICIENT_SIZE; i++)
	{
//		ys.i[i]=xs[offset];
//
//		setUvector8Element(ys, i, xs[offset]);
//		ts.i[i]=0;
//		setUvector8Element(ts, i, 0);

		ys.i0 = xs[offset];
		offset += permutationStride;
		ys.i1 = xs[offset];
		offset += permutationStride;
		ys.i2 = xs[offset];
		offset += permutationStride;
		ys.i3 = xs[offset];
		offset += permutationStride;
		ys.i4 = xs[offset];
		offset += permutationStride;
		ys.i5 = xs[offset];
		offset += permutationStride;
		ys.i6 = xs[offset];
		offset += permutationStride;
		ys.i7 = xs[offset];

		ts.i0 = 0;
		ts.i1 = 0;
		ts.i2 = 0;
		ts.i3 = 0;
		ts.i4 = 0;
		ts.i5 = 0;
		ts.i6 = 0;
		ts.i7 = 0;
	}

	j = 8 - sn;
//	usfixn64 offset = j * (permutationStride);
//	usfixn64 offset = (8-sn) * (permutationStride);
//	for (i = 0; i < sn; i++)
//	{
//		ts[i] = xs[j++];
//	}
	for (i = 0; i < sn; i++)
	{
//		ts[i] = xs[offset];
//		ts.i[i] = xs[offset];

//		ts.i[i] = ys.i[j];
		setUvector8Element (ts, i, getUvector8Element (ys, j));
		j++;
//		offset += permutationStride;
	}

//	offset = (7 - sn) * (permutationStride);
	for (i = 7 - sn; i >= 0; i--)
	{
//		xs[i + sn] = xs[i];
//		ys[i + sn] = xs[offset];
//		ys.i[i + sn] = xs[offset];

//		ys.i[i + sn] = ys.i[i];
		j = i + sn;
		setUvector8Element (ys, j, getUvector8Element (ys, i));

//		offset -= permutationStride;
	}
	for (i = 0; i < sn; i++)
	{
////		xs[i] = 0;
//		ys.i[i] = 0;
		setUvector8Element (ys, i, 0);
	}
//	memset(ys.i, 0x00, sn*sizeof(usfixn64));
//	device_p3_bigSub(ys, ts, xs);
//	device_bigSub_plain(&ys.i[0], &ts.i[0]);
//	device_bigSub_plain_1(&ys.i[0], &ts.i[0]);
//	device_bigSub_plain_1(ys.i, ts.i);
//	device_bigSub_plain_1(ys.i, ts.i);
	device_p3_bigSub_uVector_plain_2_local_add (ys, ts);
//	device_bigPrimeSub_permutated_3(xs,ts, permutationStride);
	offset = 0;
//#pragma unroll COEFFICIENT_SIZE
//	for (i = 0; i < 8; i++)
//	{
////		xs[offset] = ys[i];
////		xs[offset] = ys.i[i];
//		xs[offset] = getUvector8Element(ys, i);
//		offset += (permutationStride);
//	}

//#pragma unroll COEFFICIENT_SIZE
//	for (i = 0; i < 8; i++)
	{
//		xs[offset] = ys[i];
//		xs[offset] = ys.i[i];
		xs[offset] = ys.i0;
		offset += (permutationStride);
		xs[offset] = ys.i1;
		offset += (permutationStride);
		xs[offset] = ys.i2;
		offset += (permutationStride);
		xs[offset] = ys.i3;
		offset += (permutationStride);
		xs[offset] = ys.i4;
		offset += (permutationStride);
		xs[offset] = ys.i5;
		offset += (permutationStride);
		xs[offset] = ys.i6;
		offset += (permutationStride);
		xs[offset] = ys.i7;
//														offset += (permutationStride);
	}
}

/**********************************************/

__device__ void
device_p3_shiftRight_uConstArray8 (uConstArray8_align8 & x, usfixn64 & tmp)
{
//	tmp = x.i[7];
//	x.i[7] = x.i[6];
//	x.i[6] = x.i[5];
//	x.i[5] = x.i[4];
//	x.i[4] = x.i[3];
//	x.i[3] = x.i[2];
//	x.i[2] = x.i[1];
//	x.i[1] = x.i[0];
//	x.i[0] = tmp;
	//device_negate_plain(tmp);
	asm("{\n\t"
		".reg .u64 t0 ;\n\t"
		"mov.u64 t0,%7;\n\t"
		"mov.u64 %7,%6;\n\t"
		"mov.u64 %6,%5;\n\t"
		"mov.u64 %5,%4;\n\t"
		"mov.u64 %4,%3;\n\t"
		"mov.u64 %3,%2;\n\t"
		"mov.u64 %2,%1;\n\t"
		"mov.u64 %1,%0;\n\t"
		"mov.u64 %0,t0;\n\t"
		"}"
		:"+l"(x.i[0]),"+l"(x.i[1]),"+l"(x.i[2]),"+l"(x.i[3]),"+l"(x.i[4]),"+l"(x.i[5]),"+l"(x.i[6]),"+l"(x.i[7])
		:);
}

/**********************************************/
__device__ void
device_p3_bigPrimeAdd_correct (usfixn64 *xm, usfixn64 *ym, usfixn64 *um)
{
	unsigned short c = 0;
	short pos1;
	usfixn64 num1, num2;

//	usfixn64 um[8];

	num1 = 0;
	num2 = R - 1;
	short i;
//	usfixn64 tmp = 0;

	for (i = 0; i < 8; i++)
	{
		num1 = xm[i] + ym[i] + c;
		if (num1 < xm[i] || num1 < ym[i]) //there is overflow/truncation
		{
			um[i] = num1 + RC;
			c = 1;
		}
		else if (num1 >= R)
		{
			c = 1;
			um[i] = num1 - R;
		}
		else
		{
			um[i] = num1;
			c = 0;
		}
	}
	if (c > 0)
	{
		pos1 = -1;
		for (i = 0; i < 8; i++)
		{
			if (um[i] != 0)
			{
				pos1 = i;
				break;
			}
		}
		if (pos1 >= 0)
		{
			for (i = 0; i < pos1; i++)
			{
				um[i] = num2;
			}
			um[pos1]--;
		}
		else
		{
			um[0] = ULMAX;
			for (i = 1; i < 8; i++)
			{
				um[i] = 0;
			}
		}
	}
}

/**********************************************/
__device__ __inline__ void
device_p3_bigPrimeAdd_plain (usfixn64 *__restrict__ xm, usfixn64 *__restrict__ ym)
{
	unsigned short c = 0;
	short pos1;
	usfixn64 num1, num2;

//	usfixn64 um[8];

	num1 = 0;
	num2 = R - 1;
	short i;
//	usfixn64 tmp = 0;

	for (i = 0; i < 8; i++)
	{
		num1 = xm[i] + ym[i] + c;
		if (num1 < xm[i] || num1 < ym[i]) //there is overflow/truncation
		{
			xm[i] = num1 + RC;
			c = 1;
		}
		else if (num1 >= R)
		{
			c = 1;
			xm[i] = num1 - R;
		}
		else
		{
			xm[i] = num1;
			c = 0;
		}
	}
	if (c > 0)
	{
		pos1 = -1;
		for (i = 0; i < 8; i++)
		{
			if (xm[i] != 0)
			{
				pos1 = i;
				break;
			}
		}
		if (pos1 >= 0)
		{
			for (i = 0; i < pos1; i++)
			{
				xm[i] = num2;
			}
			xm[pos1]--;
		}
		else
		{
			xm[0] = ULMAX;
			for (i = 1; i < 8; i++)
			{
				xm[i] = 0;
			}
		}
	}
}

/**********************************************/
__device__ __inline__ void
device_p3_bigPrimeAdd_plain_inPlace (usfixn64 * __restrict__ xm,
	usfixn64 * __restrict__ ym)
{
	unsigned short c = 0;

	short pos1;
	usfixn64 num1, num2;

//	num1 = 0;
//	num2 = R - 1;
	short i;
	for (i = 0; i < 8; i++)
	{
		num1 = xm[i] + ym[i] + c;
		if (num1 < xm[i] || num1 < ym[i]) //there is overflow/truncation
		{
			xm[i] = num1 + RC;
			c = 1;
		}
		else if (num1 >= R)
		{
			c = 1;
			xm[i] = num1 - R;
		}
		else
		{
			xm[i] = num1;
			c = 0;
		}
	}

	if (c > 0)
	{
		pos1 = -1;
		if (xm[0] != 0 && pos1 == -1)
		{
			pos1 = 0;
			xm[0]--;
			return;
		}
		if (xm[1] != 0 && pos1 == -1)
		{
			pos1 = 1;
			xm[0] = R_MINUS_ONE;
			xm[1]--;
			return;
		}
		if (xm[2] != 0 && pos1 == -1)
		{
			pos1 = 2;
			xm[0] = R_MINUS_ONE;
			xm[1] = R_MINUS_ONE;
			xm[2]--;
			return;
		}
		if (xm[3] != 0 && pos1 == -1)
		{
			pos1 = 3;
			xm[0] = R_MINUS_ONE;
			xm[1] = R_MINUS_ONE;
			xm[2] = R_MINUS_ONE;
			xm[3]--;
			return;
		}
		if (xm[4] != 0 && pos1 == -1)
		{
			pos1 = 4;
			xm[0] = R_MINUS_ONE;
			xm[1] = R_MINUS_ONE;
			xm[2] = R_MINUS_ONE;
			xm[3] = R_MINUS_ONE;
			xm[4]--;
			return;
		}
		if (xm[5] != 0 && pos1 == -1)
		{
			pos1 = 5;
			xm[0] = R_MINUS_ONE;
			xm[1] = R_MINUS_ONE;
			xm[2] = R_MINUS_ONE;
			xm[3] = R_MINUS_ONE;
			xm[4] = R_MINUS_ONE;
			xm[5]--;
			return;
		}
		if (xm[6] != 0 && pos1 == -1)
		{
			pos1 = 6;
			xm[0] = R_MINUS_ONE;
			xm[1] = R_MINUS_ONE;
			xm[2] = R_MINUS_ONE;
			xm[3] = R_MINUS_ONE;
			xm[4] = R_MINUS_ONE;
			xm[5] = R_MINUS_ONE;
			xm[6]--;
			return;
		}

		if (xm[7] != 0 && pos1 == -1)
		{
			pos1 = 7;
			xm[0] = R_MINUS_ONE;
			xm[1] = R_MINUS_ONE;
			xm[2] = R_MINUS_ONE;
			xm[3] = R_MINUS_ONE;
			xm[4] = R_MINUS_ONE;
			xm[5] = R_MINUS_ONE;
			xm[6] = R_MINUS_ONE;
			xm[7]--;
			return;
		}
//else (c>0) but (pos ==-1)
		{
			xm[0] = ULMAX;
			xm[1] = 0;
			xm[2] = 0;
			xm[3] = 0;
			xm[4] = 0;
			xm[5] = 0;
			xm[6] = 0;
			xm[7] = 0;
		}
	}
}

/**********************************************/
__device__ __inline__ void
device_p3_bigPrimeAdd_plain_ptx_v0 (usfixn64 * __restrict__ xm,
	usfixn64 * __restrict__ ym)
{
//	unsigned short c = 0;

	usfixn16 c = 0;
//	short pos1;
	usfixn32 pos1;
	usfixn64 num1, num2;
	usfixn32 bitsFlag = 0x0;
//	num1 = 0;
//	num2 = R - 1;
	short i;
	short bit;
	for (i = 0; i < 8; i++)
	{
		num1 = xm[i] + ym[i] + c;
		if (num1 < xm[i] || num1 < ym[i]) //there is overflow/truncation
		{
			xm[i] = num1 + RC;
			c = 1;
		}
		else if (num1 >= R)
		{
			c = 1;
			xm[i] = num1 - R;
		}
		else
		{
			xm[i] = num1;
			c = 0;
		}
		bit = (xm[i] > 0);
		bitsFlag |= (bit << i);
	}

	asm("brev.b32 %0,%0;":"+r"(bitsFlag):);
	asm("bfind.u32 %0, %1;":"=r"(pos1):"r"(bitsFlag));
	if (pos1 == 0xFFFFFFFF)
		pos1 = -1;
	else
		pos1 = 31 - pos1;

	if (c > 0)
	{
		if (pos1 == -1)
		{
			xm[0] = ULMAX;
			xm[1] = 0;
			xm[2] = 0;
			xm[3] = 0;
			xm[4] = 0;
			xm[5] = 0;
			xm[6] = 0;
			xm[7] = 0;
		}
		else
		{
			xm[pos1]--;
			for (i = 0; i < pos1; i++)
				xm[i] = R_MINUS_ONE;
		}
	}
}

/**********************************************/
__device__ void
device_p3_bigPrimeAdd_permutated (const usfixn64 *xm, const usfixn64 * ym, usfixn64 *um,
	const usfixn64 idx, const short permutationStride)
{
	unsigned short c = 0;
	usfixn64 num1, num2;

	num1 = 0;
	num2 = R - 1;
	short i;
//	usfixn64 pos = threadIdx.x;

	short offset = 0;
	short pos1;
//	usfixn64 tmp = 0;

	offset = 0;
//#pragma unroll COEFFICIENT_SIZE
	for (i = 0; i < 8; i++)
	{
		//		num1 = xm[i] + ym[i] + c;
//		num1 = xm[pos] + ym[pos] + c;
		num1 = xm[idx + offset] + ym[idx + offset] + c;
		if (num1 < xm[idx + offset] || num1 < ym[idx + offset])	//there is overflow/truncation
		{
			um[idx + offset] = num1 + RC;
			c = 1;
		}
		else if (num1 >= R)
		{
			c = 1;
			um[idx + offset] = num1 - R;
		}
		else
		{
			um[idx + offset] = num1;
			c = 0;
		}
		offset += permutationStride;
	}

	offset = 0;
	if (c > 0)
	{
		pos1 = -1;
		for (i = 0; i < 8; i++)
		{
			if (um[idx + offset] != 0)
			{
				pos1 = i;
				break;
			}
			offset += permutationStride;
		}
		if (pos1 >= 0)	// shouldn't it be >0?
		{
			offset = 0;
			for (i = 0; i < pos1; i++)
			{
				um[idx + offset] = num2;
				offset += permutationStride;
			}
			offset = pos1 * permutationStride;
			um[idx + offset]--;
//			um[pos1*permutationStride+idx]--;
		}
		else
		{
//			um[0] = ULMAX;
			um[idx] = ULMAX;
			offset = permutationStride;
			for (i = 1; i < 8; i++)
			{
//				um[i] = 0;
				um[idx + offset] = 0;
			}
		}
	}

//	offset=0;
////	permutationStride=32;
//	for(i=0; i<8; i++)
//	{
////		um[idx+offset]=idx;
////		um[idx+offset]=blockIdx.x;
////		um[idx+offset]=111;
//		offset+=permutationStride;
//	}
}

/**********************************************/
__device__ void
device_p3_bigPrimeAdd_permutated_ptx_v0 (usfixn64 * xm, usfixn64 *ym,
	const usfixn64 permutationStride)
{
	unsigned short c = 0;
	usfixn64 num1 = 0;
	short i;
//	usfixn64 pos = threadIdx.x;

	usfixn64 offset = 0;
//	short posAdd = -1;
	usfixn32 posAdd = -1;
	usfixn32 bit;
	usfixn32 bitsFlag;
//	usfixn64 tmp = 0;

	offset = 0;
//#pragma unroll COEFFICIENT_SIZE
	for (i = 0; i < 8; i++)
	{
//		num1 = xm[i] + ym[i] + c;
//		num1 = xm[pos] + ym[pos] + c;
		num1 = xm[offset] + ym[offset] + c;
		if (num1 < xm[offset] || num1 < ym[offset])	//there is overflow/truncation
		{
//			xm[offset] = num1 + RC;
			num1 += RC;
			c = 1;
		}
		else if (num1 >= R)
		{
			c = 1;
//			xm[offset] = num1 - R;
			num1 -= R;
		}
		else
		{
//			xm[offset] = num1;
			c = 0;
		}
//		posAdd = (posAdd == -1 && num1 > 0) * i;

		xm[offset] = num1;
		bit = (num1 > 0);
		bitsFlag |= (bit << i);
		offset += permutationStride;
	}

	asm("brev.b32 %0,%0;":"+r"(bitsFlag):);
	asm("bfind.u32 %0, %1;":"=r"(posAdd):"r"(bitsFlag));
	if (posAdd == 0xFFFFFFFF)
		posAdd = -1;
	else
		posAdd = 31 - posAdd;

	offset = 0;

	offset = 0;
	if (c > 0)
	{

		if (posAdd == -1)
		{
			xm[offset] = ULMAX;
			offset += permutationStride;
			xm[offset] = 0;
			offset += permutationStride;
			xm[offset] = 0;
			offset += permutationStride;
			xm[offset] = 0;
			offset += permutationStride;
			xm[offset] = 0;
			offset += permutationStride;
			xm[offset] = 0;
			offset += permutationStride;
			xm[offset] = 0;
			offset += permutationStride;
			xm[offset] = 0;

		}
		else
		{
			for (i = 0; i < posAdd; i++)
			{
				xm[offset] = R_MINUS_ONE;
				offset += permutationStride;
			}
			xm[offset]--;
		}
	}
}

/**********************************************/
//xm = xm + ym
//ym = xm - ym
//__device__ inline void fft_base2_permutated(usfixn64 * __restrict__ xm,
//		usfixn64 * __restrict__ ym, const usfixn64 xIdx,const usfixn64 yIdx,
//		const usfixn64 permutationStride)
__device__ __inline__ void
device_p3_fft_base2_permutated (usfixn64 * xm, usfixn64 * ym,
	const usfixn64 & permutationStride)
{
	short c = 0;
	usfixn64 num1, num2;
	unsigned short c2 = 0;
	usfixn64 num3 = 0;

	num1 = 0;
	num2 = R - 1;
	short i;
//	usfixn64 pos = threadIdx.x;

	usfixn64 offset = 0;
	short pos1, pos = 0;
//	usfixn64 tmp = 0;
	usfixn64 tAdd = 0;
	offset = 0;

//#pragma unroll COEFFICIENT_SIZE
	for (i = 0; i < COEFFICIENT_SIZE; i++)
	{
		num1 = xm[offset] + ym[offset] + c;
		num3 = ym[offset] + c2;

		//addition part
		if (num1 < xm[offset] || num1 < ym[offset])	//there is overflow/truncation
		{
			num1 = num1 + RC;
			c = 1;
		}
		else if (num1 >= R)
		{
			c = 1;
			num1 = num1 - R;
		}
		else
		{
			c = 0;
		}

		//subtraction part
		if (xm[offset] < num3) //there is not enough to do subtraction
		{
			c2 = 1;
			num3 = R - num3 + xm[offset];
		}
		else
		{
			c2 = 0;
			num3 = xm[offset] - num3;
		}
		xm[offset] = num1;
		ym[offset] = num3;
		offset += permutationStride;
		__syncthreads ();
	}
//	return;

	offset = 0;
	if (c > 0)
	{
		pos1 = -1;
		for (i = 0; i < 8; i++)
		{
			if (xm[offset] != 0)
			{
				pos1 = i;
				break;
			}
			offset += permutationStride;
		}
		if (pos1 >= 0)	// shouldn't it be >0?
		{
			offset = 0;
			for (i = 0; i < pos1; i++)
			{
				xm[offset] = num2;
				offset += permutationStride;
				__syncthreads ();
			}
			offset = pos1 * permutationStride;
			xm[offset]--;
			//xm[pos1*permutationStride+idx]--;
		}
		else
		{
			//			xm[0] = ULMAX;
			xm[offset] = ULMAX;
			offset = permutationStride;
			for (i = 1; i < 8; i++)
			{
				//				xm[i] = 0;
				xm[offset] = 0;
				__syncthreads ();
			}
		}
	}

	if (c2 > 0)
	{
		offset = 0;
		pos = -1;
		for (i = 0; i < 8; i++)
		{
			if (ym[offset] <= num2)
			{
				pos = i;
				break;
			}
			offset += permutationStride;
		}

		if (pos >= 0)
		{
			offset = 0;
			for (i = 0; i < pos; i++)
			{
				ym[offset] = 0;
				offset += permutationStride;
				__syncthreads ();
			}
			offset = pos * permutationStride;
			//			um[pos]++;
			ym[offset]++;
			__syncthreads ();
		}
		else
		{

			offset = 0;
			//			um[0] = ULMAX;
			ym[offset] = ULMAX;
			offset = permutationStride;
			for (i = 1; i < 8; i++)
			{
				//				um[i] = 0;
				ym[offset] = 0;
				offset += permutationStride;
				__syncthreads ();
			}
		}
	}
}

/**********************************************/
//using vectorized data structure + offset is computed more efficiently
__device__ inline void
device_p3_fft_base2_permutated_4 (usfixn64 * __restrict__ xm, usfixn64 * __restrict__ ym,
	const usfixn64 & permutationStride)
{
	short c = 0;
	usfixn64 num1;
//	num2;
	short c2 = 0;
	usfixn64 num3 = 0;

	num1 = 0;
//	num2 = R - 1;
	short i;
//	usfixn64 pos = threadIdx.x;

	usfixn64 offset = 0;
	short posAdd = -1, posSub = -1;
//	usfixn64 tmp = 0;
	offset = 0;

//	usfixn64 xConst, yConst;
//#pragma unroll COEFFICIENT_SIZE
//	for (i = 0; i < 8; i++)
//	{
//		xConst.i[i] = xm[offset];
//		offset += permutationStride;
//	}
//
//	offset = idx;
//#pragma unroll COEFFICIENT_SIZE
//	for (i = 0; i < 8; i++)
//	{
//		yConst.i[i] = ym[offset];
//		offset += permutationStride;
//	}

//	offset=idx;
//#pragma unroll COEFFICIENT_SIZE
//	for (i = 0; i < 8; i++)
//	{
//		xConst.i[i] = xm[offset];
////	offset += permutationStride;
//		offset += STRIDE;
//	}

//	offset=idx;
//#pragma unroll COEFFICIENT_SIZE
//	for (i = 0; i < 8; i++)
//	{
//		yConst.i[i] = ym[offset];
////	offset += permutationStride;
//		offset += STRIDE;
//	}

	offset = 0;
//#pragma unroll COEFFICIENT_SIZE
	for (i = 0; i < 8; i++)
	{
//		num1 = xm[idx + offset] + ym[idx + offset] + c;
//		num3 = ym[idx + offset] + c2;
//		num1 = xm[offset] + ym[offset] + c;
//		num3 = ym[offset] + c2;
//		xConst = xm[offset];
//		yConst = ym[offset];

//		num1 = xConst + yConst + c;
//		num3 = yConst + c2;

		num1 = xm[offset] + ym[offset] + c;
		num3 = ym[offset] + c2;
		//addition part
//		if (num1 < xm[idx + offset] || num1 < ym[idx + offset])	//there is overflow/truncation
//		if (num1 < xm[offset] || num1 < ym[offset])	//there is overflow/truncation
		if (num1 < xm[offset] || num1 < ym[offset])	//there is overflow/truncation
		{
			num1 = num1 + RC;
			c = 1;
		}
		else if (num1 >= R)
		{
			c = 1;
			num1 = num1 - R;
		}
		else
		{
			c = 0;
		}

		//subtraction part
//		if (xm[idx + offset] < num3) //there is not enough to do subtraction
//		if (xm[offset] < num3) //there is not enough to do subtraction
//		{
//			c2 = 1;
////			num3 = R - num3 + xm[idx + offset];
//			num3 = R - num3 + xm[offset];
//		}
//		else
//		{
//			c2 = 0;
////			num3 = xm[idx + offset] - num3;
//			num3 = xm[offset] - num3;
//		}

		if (xm[offset] < num3) //there is not enough to do subtraction
		{
			c2 = 1;
			//			num3 = R - num3 + xm[idx + offset];
			num3 = R - num3 + xm[offset];
		}
		else
		{
			c2 = 0;
			//			num3 = xm[idx + offset] - num3;
			num3 = xm[offset] - num3;
		}
//		xm[idx + offset] = num1;
//		ym[idx + offset] = num1;
//		xm[offset] = num1;
//		ym[offset] = num1;
//		offset += permutationStride;
//		xConst=num1;
//		yConst=num3;
		posAdd = (posAdd == -1 && num1 > 0) * i;
		posSub = (posSub == -1 && num3 > 0) * i;
		xm[offset] = num1;
		ym[offset] = num3;
		offset += permutationStride;
	}

//	offset = idx;
//	if (c > 0)
//	{
////		posAdd = -1;
////		for (i = 0; i < 8; i++)
//		{
////			if (xm[idx+offset] != 0)
////			if (xm[offset] != 0)
////			if (xConst.i[i] != 0)
////			{
////				posAdd = i;
////				break;
////			}
////			offset += permutationStride;
//		}
//		offset = idx;
//		if (posAdd >= 0)	// shouldn't it be >0?
//		{
////			offset = 0;
//
//			for (i = 0; i < posAdd; i++)
//			{
////				xm[idx + offset] = num2;
//				xm[offset] = num2;
////				xConst = num2;
//				offset += permutationStride;
//			}
////			offset = pos * permutationStride;
////			xm[idx + offset]--;
//			xm[offset]--;
////			xConst.i[posAdd]--;
//			//xm[pos*permutationStride+idx]--;
//		}
//		else
//		{
//			//			xm[0] = ULMAX;
////			xm[idx] = ULMAX;
////			xConst.i[0] = ULMAX;
//			xm[offset] = ULMAX;
//			offset += permutationStride;
////			offset = idx+permutationStride;
//#pragma unroll (COEFFICIENT_SIZE-1)
//			for (i = 1; i < 8; i++)
//			{
//				//				xm[i] = 0;
////				xm[idx + offset] = 0;
//				xm[offset] = 0;
////				xConst.i[i] = 0;
//				offset+=permutationStride;
//			}
//		}
//	}

	offset = 0;
	if (c > 0 && posAdd >= 0)
	{
		for (i = 0; i < posAdd; i++)
		{
//			xm[offset] = R_MINUS_ONE;
			xm[offset] = R - 1;
			offset += permutationStride;
		}
		xm[offset]--;

	}
	else if (c > 0 && posAdd == -1)
	{
		xm[offset] = ULMAX;
		offset += permutationStride;

//#pragma unroll (COEFFICIENT_SIZE-1)
		for (i = 1; i < 8; i++)
		{
			xm[offset] = 0;
			offset += permutationStride;
		}
	}
//
//	if (c2 > 0)
//	{
////		offset = 0;
////		offset = idx;
////		posSub = -1;
////		for (i = 0; i < 8; i++)
////		{
//////			if (ym[idx + offset] < num2)
////			if (yConst.i[i] < num2)
////			{
////				posSub = i;
////				break;
////			}
//////			offset += permutationStride;
////		}
//
//		offset = idx;
//		if (posSub >= 0)
//		{
////			offset = 0;
//			for (i = 0; i < posSub; i++)
//			{
////				ym[idx + offset] = 0;
//				ym[offset] = 0;
//				offset += permutationStride;
////				yConst.i[i] = 0;
//			}
////			offset = pos * permutationStride;
//			//			um[pos]++;
////			ym[idx + offset]++;
////			yConst.i[posSub]++;
//			ym[offset]++;
//		}
//		else
//		{
//			//			um[0] = ULMAX;
////			ym[idx] = ULMAX;
////			yConst.i[0] = ULMAX;
//			ym[offset] = ULMAX;
//			offset += permutationStride;
////			offset = 0;
//#pragma unroll (COEFFICIENT_SIZE-1)
//			for (i = 1; i < 8; i++)
//			{
//				//				um[i] = 0;
////				ym[idx + offset]=0;
////				yConst.i[i] = 0;
//				ym[offset] = 0;
//				offset += permutationStride;
//			}
//		}
//	}

//		offset = 0;
//		offset = idx;
//		posSub = -1;
//		for (i = 0; i < 8; i++)
//		{
////			if (ym[idx + offset] < num2)
//			if (yConst.i[i] < num2)
//			{
//				posSub = i;
//				break;
//			}
////			offset += permutationStride;
//		}

	offset = 0;
	if (c2 > 0 && posSub >= 0)
	{
//			offset = 0;
		for (i = 0; i < posSub; i++)
		{
//				ym[idx + offset] = 0;
			ym[offset] = 0;
			offset += permutationStride;
//				yConst.i[i] = 0;
		}
//			offset = pos * permutationStride;
		//			um[pos]++;
//			ym[idx + offset]++;
//			yConst.i[posSub]++;
		ym[offset]++;
	}
	else if (c2 > 0 && posSub == -1)
	{
		//			um[0] = ULMAX;
//			ym[idx] = ULMAX;
//			yConst.i[0] = ULMAX;
		ym[offset] = ULMAX;
		offset += permutationStride;
//			offset = 0;
//#pragma unroll (COEFFICIENT_SIZE-1)
		for (i = 1; i < 8; i++)
		{
			//				um[i] = 0;
//				ym[idx + offset]=0;
//				yConst.i[i] = 0;
			ym[offset] = 0;
			offset += permutationStride;
		}
	}

//	offset = idx;
//	#pragma unroll COEFFICIENT_SIZE
//		for (i = 0; i < 8; i++)
//		{
//			xm[offset] = xConst.i[i];
////			ym[offset] = yConst.i[i];
////			offset+=permutationStride;
//			offset+=STRIDE;
//		}

//		offset = idx;
//	#pragma unroll COEFFICIENT_SIZE
//		for (i = 0; i < 8; i++)
//		{
//			ym[offset] = yConst.i[i];
////			offset+=permutationStride;
//			offset+=STRIDE;
//		}

}

/**********************************************/
__device__ inline void
device_p3_fft_base2_permutated_7 (usfixn64 * __restrict__ xm, usfixn64 * __restrict__ ym,
	const short shNo, const usfixn64 permutationStride)
{
	short c = 0;
	usfixn64 num1;
//	num2;
	unsigned short c2 = 0;
	usfixn64 num3 = 0;

	num1 = 0;
//	num2 = R - 1;
	short i;
//	usfixn64 pos = threadIdx.x;

	usfixn64 offset = 0;
	short posAdd = -1, posSub = -1;
//	usfixn64 tmp = 0;
	offset = 0;

//	usfixn64 xConst, yConst;
//#pragma unroll COEFFICIENT_SIZE
//	for (i = 0; i < 8; i++)
//	{
//		xConst.i[i] = xm[offset];
//		offset += permutationStride;
//	}
//
//	offset = idx;
//#pragma unroll COEFFICIENT_SIZE
//	for (i = 0; i < 8; i++)
//	{
//		yConst.i[i] = ym[offset];
//		offset += permutationStride;
//	}

//	offset=idx;
//#pragma unroll COEFFICIENT_SIZE
//	for (i = 0; i < 8; i++)
//	{
//		xConst.i[i] = xm[offset];
////	offset += permutationStride;
//		offset += STRIDE;
//	}

//	offset=idx;
//#pragma unroll COEFFICIENT_SIZE
//	for (i = 0; i < 8; i++)
//	{
//		yConst.i[i] = ym[offset];
////	offset += permutationStride;
//		offset += STRIDE;
//	}

	device_p3_cyclicShift_permutated_5_local_add (ym, shNo, permutationStride);
	offset = 0;
//#pragma unroll COEFFICIENT_SIZE
	for (i = 0; i < COEFFICIENT_SIZE; i++)
	{
//		num1 = xm[idx + offset] + ym[idx + offset] + c;
//		num3 = ym[idx + offset] + c2;
//		num1 = xm[offset] + ym[offset] + c;
//		num3 = ym[offset] + c2;
//		xConst = xm[offset];
//		yConst = ym[offset];

//		num1 = xConst + yConst + c;
//		num3 = yConst + c2;

//		num1 = xm[offset] + ym[offset] + c;
//		num3 = ym[offset] + c2;

		num1 = xm[offset];		// +ym[offset] + c;
		num3 = ym[offset]; //+ c2;
		num1 = num1 + c + num3;
		num3 += c2;
		//addition part
//		if (num1 < xm[idx + offset] || num1 < ym[idx + offset])	//there is overflow/truncation
//		if (num1 < xm[offset] || num1 < ym[offset])	//there is overflow/truncation
		if (num1 < xm[offset] || num1 < ym[offset])	//there is overflow/truncation
		{
			num1 = num1 + RC;
			c = 1;
		}
		else if (num1 >= R)
		{
			c = 1;
			num1 = num1 - R;
		}
		else
		{
			c = 0;
		}

		//subtraction part
//		if (xm[idx + offset] < num3) //there is not enough to do subtraction
//		if (xm[offset] < num3) //there is not enough to do subtraction
//		{
//			c2 = 1;
////			num3 = R - num3 + xm[idx + offset];
//			num3 = R - num3 + xm[offset];
//		}
//		else
//		{
//			c2 = 0;
////			num3 = xm[idx + offset] - num3;
//			num3 = xm[offset] - num3;
//		}

		if (xm[offset] < num3) //there is not enough to do subtraction
		{
			c2 = 1;
			//			num3 = R - num3 + xm[idx + offset];
			num3 = R - num3 + xm[offset];
		}
		else
		{
			c2 = 0;
			//			num3 = xm[idx + offset] - num3;
			num3 = xm[offset] - num3;
		}
//		xm[idx + offset] = num1;
//		ym[idx + offset] = num1;
//		xm[offset] = num1;
//		ym[offset] = num1;
//		offset += permutationStride;
//		xConst=num1;
//		yConst=num3;
		xm[offset] = num1;
		posAdd = (posAdd == -1 && num1 > 0) * i;
		posSub = (posSub == -1 && num3 > 0) * i;
		ym[offset] = num3;
		offset += permutationStride;
	}

//	offset = idx;
//	if (c > 0)
//	{
////		posAdd = -1;
////		for (i = 0; i < 8; i++)
//		{
////			if (xm[idx+offset] != 0)
////			if (xm[offset] != 0)
////			if (xConst.i[i] != 0)
////			{
////				posAdd = i;
////				break;
////			}
////			offset += permutationStride;
//		}
//		offset = idx;
//		if (posAdd >= 0)	// shouldn't it be >0?
//		{
////			offset = 0;
//
//			for (i = 0; i < posAdd; i++)
//			{
////				xm[idx + offset] = num2;
//				xm[offset] = num2;
////				xConst = num2;
//				offset += permutationStride;
//			}
////			offset = pos * permutationStride;
////			xm[idx + offset]--;
//			xm[offset]--;
////			xConst.i[posAdd]--;
//			//xm[pos*permutationStride+idx]--;
//		}
//		else
//		{
//			//			xm[0] = ULMAX;
////			xm[idx] = ULMAX;
////			xConst.i[0] = ULMAX;
//			xm[offset] = ULMAX;
//			offset += permutationStride;
////			offset = idx+permutationStride;
//#pragma unroll (COEFFICIENT_SIZE-1)
//			for (i = 1; i < 8; i++)
//			{
//				//				xm[i] = 0;
////				xm[idx + offset] = 0;
//				xm[offset] = 0;
////				xConst.i[i] = 0;
//				offset+=permutationStride;
//			}
//		}
//	}

	offset = 0;
	if (c > 0 && posAdd >= 0)
	{
		for (i = 0; i < posAdd; i++)
		{
			xm[offset] = R_MINUS_ONE;
			offset += permutationStride;
		}
		xm[offset]--;
	}

	if (c > 0 && posAdd == -1)
	{
		xm[offset] = ULMAX;
		offset += permutationStride;

//#pragma unroll (COEFFICIENT_SIZE-1)
		for (i = 1; i < COEFFICIENT_SIZE; i++)
		{
			xm[offset] = 0;
			offset += permutationStride;
		}
	}
//
//	if (c2 > 0)
//	{
////		offset = 0;
////		offset = idx;
////		posSub = -1;
////		for (i = 0; i < 8; i++)
////		{
//////			if (ym[idx + offset] < num2)
////			if (yConst.i[i] < num2)
////			{
////				posSub = i;
////				break;
////			}
//////			offset += permutationStride;
////		}
//
//		offset = idx;
//		if (posSub >= 0)
//		{
////			offset = 0;
//			for (i = 0; i < posSub; i++)
//			{
////				ym[idx + offset] = 0;
//				ym[offset] = 0;
//				offset += permutationStride;
////				yConst.i[i] = 0;
//			}
////			offset = pos * permutationStride;
//			//			um[pos]++;
////			ym[idx + offset]++;
////			yConst.i[posSub]++;
//			ym[offset]++;
//		}
//		else
//		{
//			//			um[0] = ULMAX;
////			ym[idx] = ULMAX;
////			yConst.i[0] = ULMAX;
//			ym[offset] = ULMAX;
//			offset += permutationStride;
////			offset = 0;
//#pragma unroll (COEFFICIENT_SIZE-1)
//			for (i = 1; i < 8; i++)
//			{
//				//				um[i] = 0;
////				ym[idx + offset]=0;
////				yConst.i[i] = 0;
//				ym[offset] = 0;
//				offset += permutationStride;
//			}
//		}
//	}

//		offset = 0;
//		offset = idx;
//		posSub = -1;
//		for (i = 0; i < 8; i++)
//		{
////			if (ym[idx + offset] < num2)
//			if (yConst.i[i] < num2)
//			{
//				posSub = i;
//				break;
//			}
////			offset += permutationStride;
//		}

	offset = 0;
	if (c2 > 0 && posSub >= 0)
	{
//			offset = 0;
		for (i = 0; i < posSub; i++)
		{
//				ym[idx + offset] = 0;
			ym[offset] = 0;
			offset += permutationStride;
//				yConst.i[i] = 0;
		}
//			offset = pos * permutationStride;
		//			um[pos]++;
//			ym[idx + offset]++;
//			yConst.i[posSub]++;
		ym[offset]++;
	}
	if (c2 > 0 && posSub == -1)
	{
		//			um[0] = ULMAX;
//			ym[idx] = ULMAX;
//			yConst.i[0] = ULMAX;
//		ym[offset] = ULMAX;
//		offset += permutationStride;
		ym[0] = ULMAX;
		offset = permutationStride;
//			offset = 0;
//#pragma unroll (COEFFICIENT_SIZE-1)
		for (i = 1; i < COEFFICIENT_SIZE; i++)
		{
			//				um[i] = 0;
//				ym[idx + offset]=0;
//				yConst.i[i] = 0;
			ym[offset] = 0;
			offset += permutationStride;
		}
	}

//	offset = idx;
//	#pragma unroll COEFFICIENT_SIZE
//		for (i = 0; i < 8; i++)
//		{
//			xm[offset] = xConst.i[i];
////			ym[offset] = yConst.i[i];
////			offset+=permutationStride;
//			offset+=STRIDE;
//		}

//		offset = idx;
//	#pragma unroll COEFFICIENT_SIZE
//		for (i = 0; i < 8; i++)
//		{
//			ym[offset] = yConst.i[i];
////			offset+=permutationStride;
//			offset+=STRIDE;
//		}

}

/**********************************************/
//using vectorized data structure + offset is computed more efficiently
__device__ void
device_p3_fft_base2_permutated_8 (usfixn64 * __restrict__ xm, usfixn64 * __restrict__ ym,
	const short & shNo, const usfixn64 & permutationStride)
{
//	device_cyclicShift_permutated_5(ym, shNo, permutationStride);
//	device_cyclicShift_permutated_6(ym, shNo, permutationStride);
	device_p3_cyclicShift_permutated_7_local_add (ym, shNo, permutationStride);
	device_p3_fft_base2_permutated (xm, ym, permutationStride);
//	fft_base2_vector8_permutated(xm, ym, permutationStride);
}

/**********************************************/
__global__ void
kernel_p3_addition_plain (usfixn64 * xs, usfixn64 * ys, usfixn64 *parameters)
//__global__ void plain(const usfixn64 * __restrict__ xs, const usfixn64 * __restrict__  ys, usfixn64 *us, const short * __restrict__ parameters)
{
	//
	short operation = parameters[0];
	short iterations = parameters[1];
	short paddingMethod = parameters[2];
	short dynamicMemSize = parameters[3];

	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	short i;
//	, pos, pos1, t;
	unsigned short c = 0;
//	usfixn64 num1, num2;
//	usfixn64 *xd, *yd, *ud, *xm, *ym, *um;
	usfixn64 *xd, *yd, *xm, *ym;

//	num1 = 0;
//	num2 = R - 1;

	xd = (usfixn64*) ((char*) xs + tid * sizeof(usfixn64) * 8);
	yd = (usfixn64*) ((char*) ys + tid * sizeof(usfixn64) * 8);
//	ud = (usfixn64*) ((char*) us + tid * sizeof(usfixn64) * 8);

//first solution, COEFFICIENT_SIZE iterations over all digits
//	device_bigPrimeAdd_plain(xd, yd);

	device_p3_bigPrimeAdd_plain_ptx_v0 (xd, yd);

	/*
	 * second solution, going on for every 2 coefficients,
	 * checking the carry at the end in bigPrimeAdd_check,
	 * should pass carry from each 2-step function, initially c=0
	 */
//	i = 0;
//	device_smallAdd2_plain(&xd[i],&yd[i],c);
//	i += 2;
//	device_smallAdd2_plain(&xd[i],&yd[i],c);
//	i += 2;
//	device_smallAdd2_plain(&xd[i],&yd[i],c);
//	i += 2;
//	device_smallAdd2_plain(&xd[i],&yd[i],c);
//	bigPrimeAdd_check(xd,c);
	}

/**********************************************/
	__global__ void
	kernel_p3_addition_permutated (usfixn64 *xs, usfixn64 *ys, usfixn64 *parameters)
	{
		short operation = parameters[0];
		usfixn64 permutationStride = parameters[5];
		short shuffle = parameters[6];
	short padding = 0;	//parameters[?]
	usfixn64 idx;
	usfixn64 tid = (threadIdx.x + blockIdx.x * blockDim.x);

	//idx = (tid / permutationBlockSize) * 8 * permutationBlockSize + (tid % permutationBlockSize);
	//following indexing is slightly faster than above indexing

	idx = tid;
//	if (padding == 0)
	device_p3_bigPrimeAdd_permutated_ptx_v0 (&xs[idx], &ys[idx], permutationStride);

//	usfixn64 offset=tid;
//	uConstArray8_align8 x,y;
//	for(short i=0;i<8;i++)
//	{
//		x.i[i]=xs[offset];
//		y.i[i]=ys[offset];
//		offset+=permutationStride;
//	}
//	device_bigPrimeAdd_plain_inPlace_2_bits(x.i,y.i);
//
//	offset=tid;
//	for(short i=0; i<8;i++)
//	{
//		xs[offset]=x.i[i];
//		offset+=permutationStride;
//	}
}

/**********************************************/
