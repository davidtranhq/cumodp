#include "hip/hip_runtime.h"
#include <cassert>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "two_convolution_poly_mul.h"
#include <thrust/copy.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int find_largest_bit_width_of_coefficients_naive(const UnivariateMPZPolynomial& a, const UnivariateMPZPolynomial& b)
{
    auto num_bits = [](const mpz_class& x) -> int {
        return mpz_sizeinbase(x.get_mpz_t(), 2);
    };
    int largest_bit_width = 0;
    for (const mpz_class& x : a)
        largest_bit_width = std::max(largest_bit_width, num_bits(x));
    for (const mpz_class& x : b)
        largest_bit_width = std::max(largest_bit_width, num_bits(x));
    return largest_bit_width;
}

#ifndef GMP_LIMB_BITS
  // Define GMP_LIMB_BITS (32 or 64) as needed.
  #define GMP_LIMB_BITS 32
#endif

// Assume mpz_limb_t is defined appropriately.
typedef unsigned long mpz_limb_t;

// __device__ helper: compute number of bits in the limb
__device__ __forceinline__ int limb_bit_length(mpz_limb_t limb) {
#if GMP_LIMB_BITS == 32
    // For 32-bit limbs use __clz
    return limb ? (32 - __clz(limb)) : 0;
#else
    // For 64-bit limbs use __clzll
    return limb ? (64 - __clzll(limb)) : 0;
#endif
}

// First kernel: compute per-element bit width and reduce within a block.
__global__ void reduce_max_bit_width_kernel(const size_t* mpz_sizes,
                                              const mpz_limb_t* ms_limbs,
                                              int n,
                                              int* partial_max) {
    extern __shared__ int sdata[];
    int tid = threadIdx.x;
    // Use unrolling by a factor of 2 for improved load efficiency.
    int idx = blockIdx.x * blockDim.x * 2 + tid;
    int max_val = 0;

    while (idx < n) {
        int bit_width = 0;
        size_t limbs = mpz_sizes[idx];
        mpz_limb_t limb = ms_limbs[idx];
        if (limbs > 0) {
            // If coefficient is nonzero, compute its bit width.
            bit_width = (limbs - 1) * GMP_LIMB_BITS + limb_bit_length(limb);
        }
        max_val = max(max_val, bit_width);

        // Unroll a second element per thread.
        int idx2 = idx + blockDim.x;
        if (idx2 < n) {
            int bit_width2 = 0;
            size_t limbs2 = mpz_sizes[idx2];
            mpz_limb_t limb2 = ms_limbs[idx2];
            if (limbs2 > 0) {
                bit_width2 = (limbs2 - 1) * GMP_LIMB_BITS + limb_bit_length(limb2);
            }
            max_val = max(max_val, bit_width2);
        }
        idx += blockDim.x * gridDim.x * 2;
    }
    sdata[tid] = max_val;
    __syncthreads();

    // Intra-block reduction in shared memory.
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s)
            sdata[tid] = max(sdata[tid], sdata[tid + s]);
        __syncthreads();
    }
    // Final warp-level reduction (no __syncthreads needed with volatile).
    if (tid < 32) {
        volatile int* smem = sdata;
        smem[tid] = max(smem[tid], smem[tid + 32]);
        smem[tid] = max(smem[tid], smem[tid + 16]);
        smem[tid] = max(smem[tid], smem[tid + 8]);
        smem[tid] = max(smem[tid], smem[tid + 4]);
        smem[tid] = max(smem[tid], smem[tid + 2]);
        smem[tid] = max(smem[tid], smem[tid + 1]);
    }
    if (tid == 0)
        partial_max[blockIdx.x] = sdata[0];
}

// Second kernel: reduce an array of ints (partial max values) to a single maximum.
__global__ void reduce_max_kernel(const int* d_in, int n, int* d_out) {
    extern __shared__ int sdata[];
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x * 2 + tid;
    int max_val = 0;
    if (idx < n)
        max_val = d_in[idx];
    if (idx + blockDim.x < n)
        max_val = max(max_val, d_in[idx + blockDim.x]);
    sdata[tid] = max_val;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s)
            sdata[tid] = max(sdata[tid], sdata[tid + s]);
        __syncthreads();
    }
    if (tid < 32) {
        volatile int* smem = sdata;
        smem[tid] = max(smem[tid], smem[tid + 32]);
        smem[tid] = max(smem[tid], smem[tid + 16]);
        smem[tid] = max(smem[tid], smem[tid + 8]);
        smem[tid] = max(smem[tid], smem[tid + 4]);
        smem[tid] = max(smem[tid], smem[tid + 2]);
        smem[tid] = max(smem[tid], smem[tid + 1]);
    }
    if (tid == 0)
        d_out[blockIdx.x] = sdata[0];
}

// Host function: launch kernels to perform the complete reduction.
int find_largest_bit_width_of_coefficients_dev(
    const thrust::device_vector<size_t>& d_mpz_sizes,
    const thrust::device_vector<mpz_limb_t>& d_most_significant_mpz_limbs)
{
    int n = d_mpz_sizes.size();
    if (n == 0) return 0;

    // First-level reduction configuration.
    int threadsPerBlock = 256;
    // We unroll by a factor of 2.
    int blocks = (n + threadsPerBlock * 2 - 1) / (threadsPerBlock * 2);

    // Allocate a device vector for block–level partial maximums.
    thrust::device_vector<int> d_partial_max(blocks);
    const size_t* raw_mpz_sizes = thrust::raw_pointer_cast(d_mpz_sizes.data());
    const mpz_limb_t* raw_ms_limbs = thrust::raw_pointer_cast(d_most_significant_mpz_limbs.data());
    int* raw_partial_max = thrust::raw_pointer_cast(d_partial_max.data());

    size_t sharedMemSize = threadsPerBlock * sizeof(int);
    // Launch the first kernel.
    reduce_max_bit_width_kernel<<<blocks, threadsPerBlock, sharedMemSize>>>
        (raw_mpz_sizes, raw_ms_limbs, n, raw_partial_max);
    hipDeviceSynchronize();

    // Continue reducing the partial results until only one value remains.
    int s = blocks;
    while (s > 1) {
        int threads = (s < threadsPerBlock * 2) ? ((s + 1) / 2) : threadsPerBlock;
        int grid = (s + threads * 2 - 1) / (threads * 2);
        thrust::device_vector<int> d_out(grid);
        int* raw_in = thrust::raw_pointer_cast(d_partial_max.data());
        int* raw_out = thrust::raw_pointer_cast(d_out.data());
        reduce_max_kernel<<<grid, threads, threads * sizeof(int)>>>(raw_in, s, raw_out);
        hipDeviceSynchronize();
        // Swap the partial results with the output for the next iteration.
        d_partial_max.swap(d_out);
        s = grid;
    }
    int result;
    thrust::copy(d_partial_max.begin(), d_partial_max.end(), &result);
    return result;
}

int find_largest_bit_width_of_coefficients_host(const UnivariateMPZPolynomial& a, const UnivariateMPZPolynomial& b)
{
    thrust::host_vector<size_t> mpz_sizes(a.size() + b.size());

    size_t total_limbs = 0;
    for (size_t i = 0; i < a.size() + b.size(); ++i) {
        mpz_sizes[i] = mpz_size(i >= a.size() ? b[i - a.size()].get_mpz_t() : a[i].get_mpz_t());
        total_limbs += mpz_sizes[i];
    }
    // Would it be better instead to hipMemcpy each coefficient individually, so that
    // each is 256-bit aligned?
    thrust::host_vector<mp_limb_t> mpz_limbs(total_limbs);
    thrust::host_vector<mp_limb_t> most_significant_mpz_limbs(a.size() + b.size());
    for (size_t i = 0, offset = 0; i < a.size() + b.size(); ++i) {
        // A lot of read-after-write hazards here, maybe could be an optimization target
        const mpz_srcptr mpz = (i >= a.size() ? b[i - a.size()].get_mpz_t() : a[i].get_mpz_t());
        std::memcpy(mpz_limbs.data() + offset, mpz->_mp_d, mpz_sizes[i] * sizeof(mp_limb_t));
        offset += mpz_sizes[i];
        most_significant_mpz_limbs[i] = mpz_sizes[i] ? mpz_limbs[offset - 1] : 0;
    }

    thrust::device_vector<size_t> d_mpz_sizes(mpz_sizes);
    thrust::device_vector<mp_limb_t> d_mpz_limbs(mpz_limbs);
    thrust::device_vector<mp_limb_t> d_most_significant_mpz_limbs(most_significant_mpz_limbs);
    return find_largest_bit_width_of_coefficients_dev(d_mpz_sizes, d_most_significant_mpz_limbs);
}


BivariateMPZPolynomial convert_to_modular_bivariate(const UnivariateMPZPolynomial& p, const BivariateBase& base, sfixn prime)
{
    assert(base.K * base.M == base.N);
    BivariateMPZPolynomial bi(p.size() * base.K);
    const int block_size {base.M};
    const int y_terms = p.size();

    auto convert_mpz_to_modular_univariate = [&](int y_power) {
        int x_power = 0;
        mpz_srcptr raw_mpz {p[y_power].get_mpz_t()};
        size_t num_limbs {mpz_size(raw_mpz)};
        
        size_t current_block_bits = 0;
        sfixn current_block = 0;
        
        size_t x_power = 0;
        // Iterate through limbs from least significant to most significant
        for (size_t limb_idx = 0; limb_idx < num_limbs; ++limb_idx) {
            mp_limb_t limb {raw_mpz->_mp_d[limb_idx]};
            
            // Process the current limb
            size_t bits_remaining {GMP_NUMB_BITS};
            while (x_power < base.K) {
                // Determine how many bits to take
                size_t bits_to_take = std::min(block_size - current_block_bits, bits_remaining);
                
                // Extract bits and add to current block
                sfixn extracted_bits {limb & ((1UL << bits_to_take) - 1)};
                current_block |= (extracted_bits << current_block_bits);
                current_block_bits += bits_to_take;
                
                // If block is full, process it
                if (current_block_bits == block_size) {
                    bi[y_power * base.K + x_power++] = current_block % prime;
                    current_block = 0;
                    current_block_bits = 0;
                }
                
                // Shift limb and update remaining bits
                limb >>= bits_to_take;
                bits_remaining -= bits_to_take;
            }
        }
        
        // Handle any remaining bits in an incomplete block
        if (current_block_bits > 0) {
            bi[y_power * base.K + x_power++] = current_block % prime;
        }
    };

    for (int y_power = 0; y_power < y_terms; ++y_power)
        convert_mpz_to_modular_univariate(y_power);
    return bi;
}


UnivariateMPZPolynomial two_convolution_poly_mul(const UnivariateMPZPolynomial& a, const UnivariateMPZPolynomial& b)
{



    
    /*
    // Find the largest bit-width of any coefficient in a or b.
    // Note that it suffices to compare the number of GMP limbs and the most significant limb in each coefficient.
    const int largest_bit_width_of_coefficients {find_largest_bit_width_of_coefficients(a, b)};

    BivariateBase base {determine_bivariate_base(largest_bit_width_of_coefficients)};
    assert(base.K * base.M == base.N);

    // Convert the univariate polynomials a and b to bivariate polynomials.

    // assuming a machine word is 64 bit
    constexpr sfixn prime_word1 = 4179340454199820289;
    constexpr sfixn prime_word2 = 2485986994308513793;

    BivariateMPZPolynomial a_bivariate1 {convert_to_modular_bivariate(a, base, prime_word1)};
    BivariateMPZPolynomial a_bivariate2 {convert_to_modular_bivariate(a, base, prime_word2)};

    BivariateMPZPolynomial b_bivariate1 {convert_to_modular_bivariate(b, base, prime_word1)};
    BivariateMPZPolynomial b_bivariate2 {convert_to_modular_bivariate(b, base, prime_word2)};

    BivariateMPZPolynomial c_minus1 {cyclic_convolution(a_bivariate1, b_bivariate1, base, prime_word1)};
    BivariateMPZPolynomial c_minus2 {cyclic_convolution(a_bivariate2, b_bivariate2, base, prime_word2)};

    BivariateMPZPolynomial c_plus1 {negacyclic_convolution(a_bivariate1, b_bivariate1, base, prime_word1)};
    BivariateMPZPolynomial c_plus2 {negacyclic_convolution(a_bivariate2, b_bivariate2, base, prime_word2)};
    */
}
