/* This file is part of the CUMODP library

    CUMODP is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    CUMODP is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with CUMODP.  If not, see <http://www.gnu.org/licenses/>.

    Copyright: Sardar Anisul Haque <shaque4@uwo.ca>
               Xin Li <xli.software@gmail.com>
               Farnam Mansouri <mansouri.farnam@gmail.com>
               Davood Mohajerani <dmohajer@uwo.ca>
               Marc Moreno Maza  <moreno@csd.uwo.ca>
               Wei Pan <wei.pan@intel.com>
               Ning Xie <nxie6@csd.uwo.ca>
*/



#include "cumodp.h"
#include "stockham.h"
#include "list_stockham.h"
#include "subres.h"
#include "cudautils.h"
#include "inlines.h"
#include "scube.h"
#include <cstdio>

/**
 * Interface for modpn C library
 * 
 * (1) Device Query
 * (2) FFT & Multiplication
 * (3) Subresultant Chain 
 *  
 * Created at Aug 24, 2010, WP
 *
 **/

///////////////////
// Device queries 
///////////////////
#ifdef __cplusplus
extern "C" 
#endif
int is_cuda_enabled() {
    int count;
    hipGetDeviceCount(&count);
    return (count > 0);
}

#ifdef __cplusplus
extern "C" 
#endif
int num_of_cuda_devices() {
    int count;
    hipGetDeviceCount(&count);
    return count;
}

#ifdef __cplusplus
extern "C" 
#endif
int is_double_float_enabled() {
    hipDeviceProp_t deviceProp;
    // only support a single cuda device, device 0
    hipGetDeviceProperties(&deviceProp, 0);
    // Returns 9999 for both major & minor fields, 
    // if no CUDA capable devices are present.
    // If major is 1, then devices with minor >= 3 
    // support double float computations.
    // If major is 2, then all these devices do.
    if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
        return 0;
    } else if (deviceProp.major == 1){
        return (deviceProp.minor >= 3);
    } 
    return (deviceProp.major > 1);
}

#ifdef __cplusplus
extern "C" 
#endif
unsigned int global_memory_in_bytes() {
    hipDeviceProp_t deviceProp;
    // only support a single cuda device, device 0
    hipGetDeviceProperties(&deviceProp, 0);
    return deviceProp.totalGlobalMem;
}

#ifdef __cplusplus
extern "C" 
#endif
float global_memory_in_megabytes() {
    hipDeviceProp_t deviceProp;
    // only support a single cuda device, device 0
    hipGetDeviceProperties(&deviceProp, 0);
    return (float)(deviceProp.totalGlobalMem >>20);
}

#ifdef __cplusplus
extern "C"
#endif
int can_call_fftmul_uni(sfixn df, sfixn dg) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    int c0 = (deviceProp.major == 9999 && deviceProp.minor == 9999);
    int c1 = (deviceProp.major == 1 && deviceProp.minor < 3);
    if (c0 || c1) return 0;
    // mem is in bytes
    size_t mem = deviceProp.totalGlobalMem;
    sfixn B = df + dg + 1;
    sfixn b = ceiling_log2(B);
    return (floor_log2(mem/4) > b + 2);
}

/* mem-usage in words */
size_t maxmem_subres(sfixn B, sfixn d1, sfixn d2) {
    if (d1 < d2) return maxmem_subres(B, d2, d1);
    // evaluation memusage + scube size + workspace
    size_t N1 = B * (1 + d1);
    size_t N2 = B * (1 + d2);
    size_t N3 = B * d2 * (d2 + 1) / 2;
    size_t N4 = 2 * B * d1 + B + 2;
    return N1 + N2 + N3 + N4;
}

#ifdef __cplusplus
extern "C"
#endif
int can_call_subres2(sfixn dx, sfixn d1, sfixn d2) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    int c0 = (deviceProp.major == 9999 && deviceProp.minor == 9999);
    int c1 = (deviceProp.major == 1 && deviceProp.minor < 3);
    if (c0 || c1) return 0;
    // mem is in bytes
    size_t mem = deviceProp.totalGlobalMem;
    sfixn B = dx * (d1 + d2) + 1;
    sfixn b = ceiling_log2(B);
    B = (sfixn)1 << b;
    return (mem / 4 > 1.3 * maxmem_subres(B, d1, d2));
}

#ifdef __cplusplus
extern "C"
#endif
int can_call_subres3(sfixn dx, sfixn dy, sfixn d1, sfixn d2) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    int c0 = (deviceProp.major == 9999 && deviceProp.minor == 9999);
    int c1 = (deviceProp.major == 1 && deviceProp.minor < 3);
    if (c0 || c1) return 0;
    // mem is in bytes
    size_t mem = deviceProp.totalGlobalMem;
    sfixn Bx = dx * (d1 + d2) + 1;
    sfixn By = dy * (d1 + d2) + 1;
    sfixn bx = ceiling_log2(Bx);
    sfixn by = ceiling_log2(By);
    Bx = ((sfixn)1 << bx);
    By = ((sfixn)1 << by);
    return (mem / 4 > 1.3 * maxmem_subres(Bx * By, d1, d2));
}

//////////////////////////////////////////////////
// FFT and FFT based polynomial multiplications
//////////////////////////////////////////////////
#ifdef __cplusplus
extern "C" 
#endif
cumodp_err 
cumodp_fft_uni(sfixn *X, sfixn n, sfixn k, sfixn w, sfixn p)
{
    // in-place 1d fft
    stockham_host(X, n, k, w, p);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        if (DEBUG) 
	fprintf(stderr, "cumodp_fft_uni: %s.\n", hipGetErrorString(err));
        return CUMODP_FFT_ERROR;
    }
    return CUMODP_SUCCESS;
}

#ifdef __cplusplus
extern "C" 
#endif
cumodp_err 
cumodp_invfft_uni(sfixn *X, sfixn n, sfixn k, sfixn w, sfixn p)
{
    // in-place 1d inverse fft
    inverse_stockham_host(X, n, k, w, p);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        if (DEBUG) fprintf(stderr, 
            "cumodp_invfft_uni: %s.\n", hipGetErrorString(err));
        return CUMODP_FFT_ERROR;
    }
    return CUMODP_SUCCESS;
}

#ifdef __cplusplus
extern "C" 
#endif
cumodp_err
cumodp_fft_bivariate(sfixn *X, sfixn em, sfixn wm, sfixn en,
    sfixn wn, sfixn p) 
{
    bivariate_stockham_host(X, em, wm, en, wn, p);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        if (DEBUG) fprintf(stderr, 
            "cumodp_fft_bivariate: %s.\n", hipGetErrorString(err));
        return CUMODP_FFT_ERROR;
    }
    return CUMODP_SUCCESS;
}

#ifdef __cplusplus
extern "C" 
#endif
cumodp_err
cumodp_invfft_bivariate(sfixn *X, sfixn em, sfixn wm, sfixn en, 
    sfixn wn, sfixn p)
{
    inverse_bivariate_stockham_host(X, em, wm, en, wn, p);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        if (DEBUG) fprintf(stderr, 
            "cumodp_invfft_bivariate: %s.\n", hipGetErrorString(err));
        return CUMODP_FFT_ERROR;
    }
    return CUMODP_SUCCESS;
}

#ifdef __cplusplus
extern "C" 
#endif
cumodp_err 
cumodp_fftmul_uni(sfixn dh, sfixn *H, sfixn df, const sfixn *F, 
    sfixn dg, const sfixn *G, sfixn p) 
{
    stockham_poly_mul_host(dh, H, df, F, dg, G, p);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        if (DEBUG) fprintf(stderr, 
            "cumodp_fftmul_uni: %s.\n", hipGetErrorString(err));
        return CUMODP_FFT_ERROR;
    }
    return CUMODP_SUCCESS;
}

////////////////////////////////////
// Subresultant chain constructions
////////////////////////////////////
#ifdef __cplusplus
extern "C" 
#endif
sfixn cumodp_subres_chain2_coarse(sfixn *S, sfixn B, sfixn w, 
    sfixn npx, sfixn npy, const sfixn *P, 
    sfixn nqx, sfixn nqy, const sfixn *Q, sfixn p) 
{
    assert(npy < nqy);
    return subres_chain2_coarse_host(S, B, w, npx, npy, P, nqx, nqy, Q, p);   
}

#ifdef __cplusplus
extern "C" 
#endif
sfixn cumodp_subres_chain3_coarse(sfixn *S, sfixn Bx, sfixn By, 
    sfixn wx, sfixn wy, sfixn npx, sfixn npy, sfixn npz, const sfixn *P, 
    sfixn nqx, sfixn nqy, sfixn nqz, const sfixn *Q, sfixn p) 
{
    assert(npz >= nqz);
    return subres_chain3_coarse_host(S, Bx, By, wx, wy, npx, npy, npz, P, 
        nqx, nqy, nqz, Q, p);
}

#ifdef __cplusplus
extern "C" 
#endif
sfixn cumodp_subres_chain2_fine(sfixn *S, sfixn B, sfixn w, 
    sfixn npx, sfixn npy, const sfixn *P, 
    sfixn nqx, sfixn nqy, const sfixn *Q, sfixn p) 
{
    assert(npy >= nqy);
    return subres_chain2_fine_host(S, B, w, npx, npy, P, nqx, nqy, Q, p);
}

#ifdef __cplusplus
extern "C" 
#endif
sfixn 
cumodp_subres_chain3_fine(sfixn *S, sfixn Bx, sfixn By, sfixn wx, sfixn wy,
    sfixn npx, sfixn npy, sfixn npz, const sfixn *P, 
    sfixn nqx, sfixn nqy, sfixn nqz, const sfixn *Q, sfixn p) 
{
    assert(npz >= nqz);
    return subres_chain3_fine_host(S, Bx, By, wx, wy, npx, npy, npz, P, 
        nqx, nqy, nqz, Q, p);
}

///////////////////////////////////////////////////////////////////////////////
// Callback functions for keeping scube inside GPU
///////////////////////////////////////////////////////////////////////////////
#ifdef __cplusplus
extern "C" 
#endif
void *init_cuda_scube(sfixn N, const sfixn *sz_p, const sfixn *sz_q, sfixn fp) 
{   
    if (!is_double_float_enabled()) return NULL;

    scube_t *scb = new scube_t(N, sz_p, sz_q, fp);

    // set the cut-off value for the fft size
    const int MIN_SCUBE2_FFT_EXP = 10;
    const int MIN_SCUBE3_FFT_EXP = 6;
    sfixn ex = scb->get_bounds_exp(0);
    if (N == 2) {
        if (ex <= MIN_SCUBE2_FFT_EXP) {
            delete scb;
            return NULL;
        }
    }

    if (N == 3) {
        sfixn ey = scb->get_bounds_exp(1);
        if (ex <= MIN_SCUBE3_FFT_EXP || ey <= MIN_SCUBE3_FFT_EXP) {
            delete scb;
            return NULL;   
        }
    }
    return (void *) scb;
}

#ifdef __cplusplus
extern "C" 
#endif
void free_cuda_scube(void *S) {
    scube_t *scb = (scube_t *)S;
    delete scb;
}

#ifdef __cplusplus
extern "C" 
#endif
void print_cuda_scube(const void *S) {
    scube_t *scb = (scube_t *)S;
    scb->info();
}

#ifdef __cplusplus
extern "C" 
#endif
cumodp_err 
build_cuda_scube(void *S, const sfixn *sz_p, const sfixn *P, 
    const sfixn *sz_q, const sfixn *Q) 
{
    scube_t *scb = (scube_t *)S;
    int N = scb->num_of_vars();
    bool ret;
    assert(N == 2 || N == 3); 
    if (N == 2) {
        ret = scb->build_scube_data2(sz_p[0], P, sz_q[0], Q); 
    } else {
        ret = scb->build_scube_data3(sz_p[0], sz_p[1], P, sz_q[0], sz_q[1], Q); 
    }
    // scb->info();
    return (ret == true) ? CUMODP_SUCCESS : CUMODP_FAILURE;
}

#ifdef __cplusplus
extern "C"
#endif
const sfixn *interp_subres_coeff2(sfixn *nx, void *S, sfixn i, sfixn j) 
{
    scube_t *scb = (scube_t *)S;
    sfixn w = scb->get_ldeg();
    if (j > i || i >= w || j < 0) { return NULL; }
    *nx = (sfixn(1) << scb->get_bounds_exp(0));
    // scb->info();
    return scb->subres_coeff(i, j);
}

#ifdef __cplusplus
extern "C"
#endif
const sfixn *interp_subres_coeff3(sfixn *nx, sfixn *ny, void *S, 
    sfixn i, sfixn j) 
{
    scube_t *scb = (scube_t *)S;
    sfixn w = scb->get_ldeg();
    if (j > i || i >= w || j < 0) return NULL;
    *nx = (sfixn(1) << scb->get_bounds_exp(0));
    *ny = (sfixn(1) << scb->get_bounds_exp(1));
    // scb->info();
    return scb->subres_coeff(i, j);
}

/////////////// END OF FILE ///////////////////////////////////////////////////


///////////////////////////////////////////////////////////////////////////////
//BEGIN:uni_fft_tst
///////////////////////////////////////////////////////////////////////////////
int uni_fft_tst(sfixn p, sfixn k) {
    
    sfixn i;
	//, p = 469762049, k = 24, n = (1L << k), w = 37;
	sfixn n = (1L << k);
	sfixn w = primitive_root(k,p);
    sfixn *X = (sfixn *)malloc(n*sizeof(sfixn));
    for (i = 0; i < n; ++i) X[i] = i % p;
    cumodp_err err = cumodp_fft_uni(X, n, k, w, p);
    
    if (err != CUMODP_SUCCESS) {
        //fprintf(stdout, "=====================\n");
        //fprintf(stdout, "Fail to do FFT for k=%d\n",k);
        //fprintf(stdout, "=====================\n");
        free(X); return -1;
    }

    err = cumodp_invfft_uni(X, n, k, w, p);
    if (err != CUMODP_SUCCESS) {
        //fprintf(stdout, "=======================\n");
        //fprintf(stdout, "Fail to do inverse FFT for k=%d\n",k);
        //fprintf(stdout, "=======================\n");
        free(X); return -2;
    }

    for (i = 0; i < n; ++i) {
        if (X[i] != i) {
            //fprintf(stderr, "Incorrect result for fft & inverse fft\n");
            free(X); return -3;
        }
    }
    free(X);
	return 0;
}
///////////////////////////////////////////////////////////////////////////////
//END:uni_fft_tst
///////////////////////////////////////////////////////////////////////////////
